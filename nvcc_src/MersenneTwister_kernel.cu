#include "hip/hip_runtime.h"

/*****************************************************************************
 *
 * This file contains code adapted from CUDA SDK 2.3 
 *
 * The Mersenne Twister RNG kernel from the CUDA SDK, modified so that
 * instead of standalone generating numbers and storing them,
 * it is called to get next number like the standard C library rand() etc,.
 * from the device only (not a __global__ kernel any more).
 *
 * $Id: MersenneTwister_kernel.cu 3350 2010-02-18 00:32:08Z alexs $
 *****************************************************************************/

/*
 * Copyright 1993-2009 NVIDIA Corporation.  All rights reserved.
 *
 * NVIDIA Corporation and its licensors retain all intellectual property and 
 * proprietary rights in and to this software and related documentation and 
 * any modifications thereto.  Any use, reproduction, disclosure, or distribution 
 * of this software and related documentation without an express license 
 * agreement from NVIDIA Corporation is strictly prohibited.
 * 
 */

#include <stdio.h>
#include <cutil_inline.h>
#include "MersenneTwister.h"



__device__ static mt_struct_stripped ds_MT[MT_RNG_COUNT];
/* static mt_struct_stripped h_MT[MT_RNG_COUNT]; */


///////////////////////////////////////////////////////////////////////////////
//
// Initialize RNG state vecotr
//
///////////////////////////////////////////////////////////////////////////////
__device__ void InitRandomGPU(int *iState0, unsigned int mt[MT_NN])
{
  const int      tid = blockDim.x * blockIdx.x + threadIdx.x;
  int iState;

  int iRng = tid;

  //Load bit-vector Mersenne Twister parameters
  mt_struct_stripped config = ds_MT[iRng];

  //Initialize current state
  mt[0] = config.seed;
  for(iState = 1; iState < MT_NN; iState++)
    mt[iState] = (1812433253U * (mt[iState - 1] ^ (mt[iState - 1] >> 30)) + iState) & MT_WMASK;
  *iState0 = 0;
}

///////////////////////////////////////////////////////////////////////////////
//
// Get next random number in sequence, for this thread.
//
///////////////////////////////////////////////////////////////////////////////
__device__ float RandomGPU(
  int *iState,
  unsigned int mt[MT_NN]
){
    const int      tid = blockDim.x * blockIdx.x + threadIdx.x;
//    const int THREAD_N = blockDim.x * gridDim.x;

    int iState1, iStateM;
    unsigned int mti, mti1, mtiM, x;

    int iRng = tid;
    //Load bit-vector Mersenne Twister parameters
    mt_struct_stripped config = ds_MT[iRng];
    mti1 = mt[0];

    //iState1 = (iState +     1) % MT_NN
    //iStateM = (iState + MT_MM) % MT_NN
    iState1 = *iState + 1;
    iStateM = *iState + MT_MM;
    if(iState1 >= MT_NN) iState1 -= MT_NN;
    if(iStateM >= MT_NN) iStateM -= MT_NN;
    mti  = mti1;
    mti1 = mt[iState1];
    mtiM = mt[iStateM];
    
    x    = (mti & MT_UMASK) | (mti1 & MT_LMASK);
    x    =  mtiM ^ (x >> 1) ^ ((x & 1) ? config.matrix_a : 0);
    mt[*iState] = x;
    *iState = iState1;
    
    //Tempering transformation
    x ^= (x >> MT_SHIFT0);
    x ^= (x << MT_SHIFTB) & config.mask_b;
    x ^= (x << MT_SHIFTC) & config.mask_c;
    x ^= (x >> MT_SHIFT1);
    
    //Convert to (0, 1] float and return
    float newrand = ((float)x + 1.0f) / 4294967296.0f;
#ifdef __DEVICE_EMULATION__
    fprintf(stderr, "%d RandomGPU %f\n", tid, newrand);
#endif
    return newrand;
}


#ifdef UNUSED

////////////////////////////////////////////////////////////////////////////////
// Transform each of MT_RNG_COUNT lanes of NPerRng uniformly distributed 
// random samples, produced by RandomGPU(), to normally distributed lanes
// using Cartesian form of Box-Muller transformation.
// NPerRng must be even.
////////////////////////////////////////////////////////////////////////////////
#define PI 3.14159265358979f
__device__ void BoxMuller(float& u1, float& u2){
    float   r = sqrtf(-2.0f * logf(u1));
    float phi = 2 * PI * u2;
    u1 = r * __cosf(phi);
    u2 = r * __sinf(phi);
}

__global__ void BoxMullerGPU(float *d_Random, int NPerRng){
    const int      tid = blockDim.x * blockIdx.x + threadIdx.x;
    const int THREAD_N = blockDim.x * gridDim.x;

    for(int iRng = tid; iRng < MT_RNG_COUNT; iRng += THREAD_N)
        for(int iOut = 0; iOut < NPerRng; iOut += 2)
            BoxMuller(
                d_Random[iRng + (iOut + 0) * MT_RNG_COUNT],
                d_Random[iRng + (iOut + 1) * MT_RNG_COUNT]
            );
}
#endif

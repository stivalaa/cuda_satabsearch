#include "hip/hip_runtime.h"
/*****************************************************************************
 * 
 * File:    cudaSaTabsearch.cu
 * Author:  Alex Stivala
 * Created: January 2010
 *
 * $Id: cudaSaTabsearch.cu 3600 2010-05-03 07:17:03Z alexs $
 *
 * CUDA host code for simulated annealing tableau matching (discrete).
 * This is a CUDA implemenation of the FORTRAN subroutine TSAMTD.
 * Since the GPU has limited memory (and specifically, very limited
 * per block shared memory), we split the database into 'small' and
 * 'large' structures. The small ones can run on the GPU in shared memory,
 * the large ones cannot so we either have to not use shared memory
 * (OK, but a bit slower) or run them on the host.
 * When runnign on the host, we can simultaneously run the GPU and
 * host in separate threads. For multiple GPU cards, CUDA also requires
 * that there is a separate host thread for each GPU, so this program
 * is multithreaded: each thread is either for a separate GPU or for
 * running the same kernel (but compiled for host) on the host CPU.
 *
 * Usage: cudaSaTabsearch [-c] [-q dbfile] [-r restarts] < inputfile
 *
 * -c : run on host CPU not GPU card
 *
 * -q : query list mode: instead of reading query data on stdin
 *      just as in the original Fortran version tlocsd, a list
 *      of query sids to be read from the database is read on stdin (one per
 *      line),
 *      and db filenaame is specified on command
 *      line. In this mode options are assumed as LORDER=T, LTYPE=T,
 *      LSOLN=N. The output is still to stdout, but each query following
 *      immediately from the previous (can parse using the  header comment
 *      niformation lines as separators.
 *
 * -r restarts: number of restarts (iterations of cooling schedule).
 *              Should be a multiple of blocksize. Defaults to 128.
 *
 * The 'database' to search is an ASCII file of  tableaux
 * (Omega matrices) in format described in rdtabd.f.
 *
 * The results are printed to stdout as 
 *
 * name score
 *
 *
 * Both the name of the database file to read, and the actual
 * query tableau are read from stdin. 
 * The first line is the name
 * of the database file.
 * The second line is for options. There are currently 3 logical
 * options, for SSE type constraint (only allow SSEs of same type ot
 * match) and ordering constraint (disallow out of sequence order 
 * matches). The third is to output not just the scores but also solution
 * vector values.
 * They are single character logical values (T or F).
 * First is type, second is order, third is solution output,
 * separated by one space.
 *
 * The subsequent lines are a single tableau in the same format as
 * each tableau entry in the database i.e.:
 *
 * The first line of an entry is the identifier and
 * order of tableau (i.e. dimension of square array), then
 * each subsequent row is a row of the tableau, lower triangle
 * only (since it is symmetric).
 * The diagonal entries are meaningless (self-angle) in tableaux,
 * and are included instead to specify the SSE type, with
 * the following codes:
 *
 * e     beta strand
 * xa    alpha helix
 * xi    pi helix
 * xg    3_10 helix
 *
 * Width of identifier is 8 chars, blank padded on right,
 * width of order is 4 digits, blank padded on left.
 * There is a single space between identifier and order.
 * Each entry in tableau is two characters, with a space betwen
 * each on a line, and one line
 * per row of matrix.
 *
 * Following the tableau is the distance matrix.
 * Each row is a row of the distance matrix, lower triangle
 * only (since it is symmetric).
 * The diagonal entries are meaningless (self-distance)
 * and are included instead to specify the SSE type, with
 * the following codes:
 * 
 * 0.000 beta strand
 * 1.000 alpha helix
 * 2.000 pi helix
 * 3.000 3_10 helix
 * 
 * Each entry in matrix is in Angstroms format
 * F6.3 with a space between each on a line, and one line
 * per row of matrix.
 * 
 * 
 * E.g.:
 * 
 * /local/charikar/astivala/tableauxdb/astral/tableauxdistmatrixdb.ascii
 *  T T F
 * D1UBIA_    8
 * e  
 * OT e  
 * LE RT xa 
 * PD OS RD xg 
 * RT LE RT LS e  
 * LE RD LE LS OT e  
 * RT LS LS RD PE OS xg 
 * PE RT LE RD OT PE RT e  
 *  0.000 
 *  4.501  0.000 
 *  1.662 10.386  1.000 
 * 16.932 17.644  9.779  3.000 
 * 10.588 13.738 11.815 10.527  0.000 
 * 15.025 18.692 17.143 15.341  6.466  0.000 
 * 15.298 17.276 16.276 20.075 13.264 11.610  3.000 
 *  7.549 11.072 12.248 12.446  4.583  9.903 15.689  0.000 
 *
 * 
 *
 *****************************************************************************/


#include <stdlib.h>
#include <stdio.h>
#include <getopt.h>
#include <time.h>
#include <string.h>
#include <multithreading.h>
#include <cutil_inline.h>
#include "MersenneTwister.h"
#include "parsetableaux.h"
#include "cudaSaTabsearch_kernel.h"


/*****************************************************************************
 *
 * Type definitions
 *
 *****************************************************************************/

/* dbIndex_t is for the query list mode, an array of these gives for each
   query the index in the appropriate ('small' or 'large' according to the
   large flag) tableaux and distmatrix db arrays */
typedef struct dbIndex_s 
{
    bool large;  /* true if query is 'large' (>MAXDIM_GPU) structure */
    int  index;  /* index in tableaux and distmatrix db list, or 'large'
                    tableaux and distmatrix db list if large is true */
} dbIndex_t;

/* searchParams_t is a struct for parameter to tableau search functions
   dcelared as CUT_THREADROUTINE to be callable as threads */
typedef struct searchParams_s
{
    int ltype; int lorder; int lsoln; /* type,order,soln flags */
    int maxstart;           /* number of restarts */
    int maxdim;             /*dimension of tableaux, distmatrices here */
    int num_queries;        /* number of queries; 0 if not query list mode */
    int single_query_qid; /* if >=0, do only the one at this index */
    dbIndex_t *query_dbindex_list; /* if num_queries>0, the query db index */
    char qtab[MAXDIM*MAXDIM];     /* if num_queries==0, the query tableau */
    float qdmat[MAXDIM*MAXDIM];   /*                    the query distmatrix*/
    char qid[LABELSIZE+1];        /*                    the query identifier*/
    int qn;                       /*                    the query order */
    char *qssetypes;              /*                    the query SSE types*/

    int dbsize;             /* number of entries in the db */
    char *tableaux;         /* the tableaux database */
    float *distmatrices;    /* the distance matrices database */
    int   *orders;          /* orders of entries in db */
    char  *names;           /* names of entries in db */
    
} searchParams_t;


/*****************************************************************************
 *
 * Globals
 *
 *****************************************************************************/

static char dbfile[MAX_LINE_LEN];   /* database file name */
static bool use_gpu = true;   /* use the GPU */
static bool use_shared_memory = true; /* use GPU shared mem for db structs */
static char *tableaux, *large_tableaux; /* small and large tableaux */
static float *distmatrices, *large_distmatrices; /* same for dist.matrices*/
static int *orders, *large_orders; /* and for orders */
static char *names, *large_names;  /* and names */
static bool querydbmode = false;   /* use list of query ids in db */
static char *queryid_list = NULL;  /* this is the list of query ids */
static dbIndex_t *query_dbindex_list = NULL; /* and their indices in db */

static int maxstart = DEFAULT_MAXSTART; /* number of restarts */


/*****************************************************************************
 *
 * This part contains code adapted from CUDA SDK 2.3 
 *
 *****************************************************************************/
/*
 * Copyright 1993-2009 NVIDIA Corporation.  All rights reserved.
 *
 * NVIDIA Corporation and its licensors retain all intellectual property and 
 * proprietary rights in and to this software and related documentation and 
 * any modifications thereto.  Any use, reproduction, disclosure, or distribution 
 * of this software and related documentation without an express license 
 * agreement from NVIDIA Corporation is strictly prohibited.
 * 
 */




///////////////////////////////////////////////////////////////////////////////
// Common host and device function 
///////////////////////////////////////////////////////////////////////////////
//ceil(a / b)
extern "C" int iDivUp(int a, int b){
    return ((a % b) != 0) ? (a / b + 1) : (a / b);
}

//floor(a / b)
extern "C" int iDivDown(int a, int b){
    return a / b;
}

//Align a to nearest higher multiple of b
extern "C" int iAlignUp(int a, int b){
    return ((a % b) != 0) ?  (a - a % b + b) : a;
}

//Align a to nearest lower multiple of b
extern "C" int iAlignDown(int a, int b){
    return a - a % b;
}



///////////////////////////////////////////////////////////////////////////////
// Reference MT front-end
///////////////////////////////////////////////////////////////////////////////
extern "C" void initMTRef(const char *fname);
extern "C" void RandomHost(void);


///////////////////////////////////////////////////////////////////////////////
// Data configuration
///////////////////////////////////////////////////////////////////////////////
//const int    PATH_N = 24000000;
//const int N_PER_RNG = iAlignUp(iDivUp(PATH_N, MT_RNG_COUNT), 2);
//const int    RAND_N = MT_RNG_COUNT * N_PER_RNG;

const unsigned int SEED = 777;

int InitRandomNumberGenerator(void) {



    fprintf(stderr,"Loading CPU and GPU twisters configurations...\n");
/*
        const char *raw_path = cutFindFilePath("MersenneTwister.raw", argv[0]);
        const char *dat_path = cutFindFilePath("MersenneTwister.dat", argv[0]);
*/
        const char *raw_path = "data/MersenneTwister.raw";
        const char *dat_path = "data/MersenneTwister.dat"; 

        if (use_gpu) {
            loadMTGPU(dat_path);
            seedMTGPU(SEED);
        }

        initMTRef(raw_path);

    return 0;
}



/*****************************************************************************
 *
 * End of CUDA SDK 2.3 Mersenne Twister code
 *
 *****************************************************************************/

/*
 * tabsearch_host_thread - run the tableau search kernel on host CPU
 *
 * Started as a thread by cutStartThread in main
 *
 * Parameters:
 *   params - paramter block for thread. See comments on searchParams_t defn.
 *
 * Return value: None.
 *
 */
static CUT_THREADPROC tabsearch_host_thread(searchParams_t *params)
{
  /* extern declartions of host version of gpu constant memory */
  extern int c_qn_host;    // query structure size
  extern char c_qtab_host[MAXDIM*MAXDIM];  // query tableau
  extern float c_qdmat_host[MAXDIM*MAXDIM];  // query distance matrix
  extern char c_qssetypes_host[MAXDIM]; // main diagonal of c_qn


  unsigned int hTimer;
  double runtime;
  int *ssemaps;
  int i,j;
  char qid[LABELSIZE+1];
  int *scores;

  int query_count = (params->num_queries == 0 || params->single_query_qid >= 0
                     ? 1 : params->num_queries);

  hipExtent tableaux_extent = {params->maxdim, params->maxdim,
                                params->dbsize};
  hipPitchedPtr tableaux_pp = {params->tableaux, params->maxdim,
                                params->maxdim, params->dbsize};
  hipExtent distmatrices_extent = {params->maxdim*sizeof(float), 
                                    params->maxdim,
                                    params->maxdim};
  hipPitchedPtr distmatrices_pp = {params->distmatrices, 
                                    params->maxdim*sizeof(float),
                                    params->maxdim,
                                    params->maxdim};


  /* allocate space for output */
  if (!(scores = (int *)malloc(params->dbsize*sizeof(int))))
  {
    fprintf(stderr, "malloc scores failed\n");
    return;
  }
  if (!(ssemaps = (int *)malloc(params->dbsize*MAXDIM*sizeof(int))))
  {
    fprintf(stderr, "malloc ssemaps failed\n");
    return;
  }

  for (int qi = 0; qi < query_count; qi++)
  {
    if (params->query_dbindex_list)
    {
      dbIndex_t *dbindex_entry =  params->single_query_qid >= 0 ? 
        &params->query_dbindex_list[params->single_query_qid] :
        &params->query_dbindex_list[qi];
      int qdbi = dbindex_entry->index;

      if (dbindex_entry->large) /* query in 'large' struct db */
      {
        strncpy(qid, large_names+qdbi*(LABELSIZE+1), LABELSIZE);
        c_qn_host = large_orders[qdbi];
        memcpy(c_qtab_host, large_tableaux+qdbi*MAXDIM*MAXDIM,
               MAXDIM*MAXDIM*sizeof(char));
        memcpy(c_qdmat_host, large_distmatrices+qdbi*MAXDIM*MAXDIM,
               MAXDIM*MAXDIM*sizeof(float));
        /* NB the query in constant memory is MAXDIM not MAXDIM_GPU 
           since constant memory larger than shared memory. */
        // set the qssetypes vector as main diagonal of the query tableau
        for (i = 0; i < c_qn_host; i++)
          c_qssetypes_host[i] = (large_tableaux+qdbi*MAXDIM*MAXDIM)[INDEX2D(i,i,MAXDIM,MAXDIM)];
      }
      else /* query in 'small' struct db */
      {
        strncpy(qid, names+qdbi*(LABELSIZE+1), LABELSIZE);
        c_qn_host = orders[qdbi];
        
        /* NB the query in constant memory is MAXDIM not MAXDIM_GPU 
           since constant memory larger than shared memory.
           This means we need to reformat the matrices into the larger 
             size if they are in the smaller class */
        for (i = 0; i < orders[qdbi]; i++)
        {
          for (j = i + 1; j < orders[qdbi]; j++)
          {
            char tabcode = (tableaux+qdbi*MAXDIM_GPU*MAXDIM_GPU)[INDEX2D(i,j,MAXDIM_GPU,MAXDIM_GPU)];
            c_qtab_host[INDEX2D(i,j,MAXDIM,MAXDIM)] = tabcode;
            c_qtab_host[INDEX2D(j,i,MAXDIM,MAXDIM)] = tabcode;
            float dist = (distmatrices+qdbi*MAXDIM_GPU*MAXDIM_GPU)[INDEX2D(i,j,MAXDIM_GPU,MAXDIM_GPU)];
            c_qdmat_host[INDEX2D(i,j,MAXDIM,MAXDIM)] = dist;
            c_qdmat_host[INDEX2D(j,i,MAXDIM,MAXDIM)] = dist;
          }
        }
        // set the qssetypes vector as main diagonal of the query tableau
        for (i = 0; i < c_qn_host; i++)
          c_qssetypes_host[i] = (tableaux+qdbi*MAXDIM_GPU*MAXDIM_GPU)[INDEX2D(i,i,MAXDIM_GPU,MAXDIM_GPU)];
      }
    }
    else
    {
      strncpy(qid, params->qid, LABELSIZE);
      c_qn_host = params->qn;
      memcpy(c_qtab_host, params->qtab, sizeof(c_qtab_host));
      memcpy(c_qdmat_host, params->qdmat, sizeof(c_qdmat_host));
      memcpy(c_qssetypes_host, params->qssetypes, sizeof(c_qssetypes_host));
    }
    
    printf("# cudaSaTabsearch LTYPE = %c LORDER = %c LSOLN = %c\n",
           params->ltype ? 'T' : 'F' , 
           params->lorder ? 'T' : 'F' , 
           params->lsoln ? 'T' : 'F');
    printf("# QUERY ID = %-8s\n", qid);
    printf("# DBFILE = %-80s\n", dbfile);
      
    fprintf(stderr, "Executing simulated annealing tableaux match kernel on host for query %s...\n", qid);
    cutilCheckError( cutCreateTimer(&hTimer) );
    cutilCheckError( cutResetTimer(hTimer) );
    cutilCheckError( cutStartTimer(hTimer) );
    sa_tabsearch_host(params->dbsize,
                      params->lorder, 
                      params->lsoln,
                      params->maxstart,
                      tableaux_pp, tableaux_extent,
                      params->orders,
                      distmatrices_pp, distmatrices_extent,
                      scores,
                      ssemaps);
    cutilCheckError( cutStopTimer(hTimer) );
    runtime = cutGetTimerValue(hTimer);
    fprintf(stderr,  "host execution time %f ms\n", runtime);
    fprintf(stderr,  "%f million iterations/sec\n", (params->dbsize * (params->maxstart * MAXITER) / (runtime/1000)) / 1.0e6);
    
    for (i = 0; i < params->dbsize; i++)
    {
      printf("%-8s  %d\n", params->names+i*(LABELSIZE+1), scores[i]);
      if (params->lsoln)
        for (int k = 0; k < c_qn_host; k++)
          if (ssemaps[i*MAXDIM + k] >= 0)
            printf("%3d %3d\n", k+1, ssemaps[i*MAXDIM + k]+1);
    }
  }
  free(scores);
  if (params->lsoln)
    free(ssemaps);
}



/*
 * copyQueryToConstantMemory() - copy the query data to device constant memory
 *
 *
 * Parameters:
 *   qi - the query index of the query to copy. 
 *        Otherwise (query_dbinex_list is NULL), these used:
 *   qn -query order
 *   qtab - query tableau  (in/out: may be set here)
 *   qdmat - query distance matrix  (in/out: may be set here)
 *   qssetypes - query SSE types vector (in/out: may be set here)
 *   qid - query id (in/out: may be set here)
 *   c_qn_symbol - name of the c_qn constant ("q_qn" or "c_qn_noshared")
 *   c_qtab_symbol - name fo the c_qtab constant
 *   c_qdmat_synmbol - name of the c_qdmat constant
 *   c_qssetypes_symbol - name of the c_qssetypes constant
 *   
 *
 * Uses the global variables query_dbindex_list, tableaux, etc.
 *
 * Return value: None.
 *
 */
static void copyQueryToConstantMemory(int qi, 
                                      int qn, char *qtab, float *qdmat,
                                      char *qssetypes, char *qid,
                                      const char *c_qn_symbol,
                                      const char *c_qtab_symbol,
                                      const char *c_qdmat_symbol,
                                      const char *c_qssetypes_symbol)
{
  unsigned int hTimer;
  cutilCheckError( cutCreateTimer(&hTimer) );
  cutilCheckError( cutResetTimer(hTimer) );
  cutilCheckError( cutStartTimer(hTimer) );
  if (query_dbindex_list)
  {
    int qdbi = query_dbindex_list[qi].index;
    if (query_dbindex_list[qi].large)
    {
      strncpy(qid, large_names+qdbi*(LABELSIZE+1), LABELSIZE);
      // set the qssetypes vector as main diagonal of the query tableau
      for (int i = 0; i < large_orders[qdbi]; i++)
        qssetypes[i] = (large_tableaux+qdbi*MAXDIM*MAXDIM)[INDEX2D(i,i,MAXDIM,MAXDIM)];
      /* copy query structure to constant memory on device */
      cutilSafeCall( hipMemcpyToSymbol(HIP_SYMBOL(c_qn_symbol), &large_orders[qdbi], sizeof(int)) );
      /* NB the query in constant memory is MAXDIM not MAXDIM_GPU 
         since constant memory larger than shared memory. */
      cutilSafeCall( hipMemcpyToSymbol(HIP_SYMBOL(c_qtab_symbol), large_tableaux+qdbi*MAXDIM*MAXDIM, MAXDIM*MAXDIM*sizeof(char)) );
      cutilSafeCall( hipMemcpyToSymbol(HIP_SYMBOL(c_qdmat_symbol), large_distmatrices+qdbi*MAXDIM*MAXDIM, MAXDIM*MAXDIM*sizeof(float)) );
      cutilSafeCall( hipMemcpyToSymbol(HIP_SYMBOL(c_qssetypes_symbol), qssetypes, MAXDIM*sizeof(char)) );
    }
    else /* query is in the 'small' structure dbase */
    {
      strncpy(qid, names+qdbi*(LABELSIZE+1), LABELSIZE);
      // set the qssetypes vector as main diagonal of the query tableau
      for (int i = 0; i < orders[qdbi]; i++)
        qssetypes[i] = (tableaux+qdbi*MAXDIM_GPU*MAXDIM_GPU)[INDEX2D(i,i,MAXDIM_GPU,MAXDIM_GPU)];
      /* copy query structure to constant memory on device */
      cutilSafeCall( hipMemcpyToSymbol(HIP_SYMBOL(c_qn_symbol), &orders[qdbi], sizeof(int)) );
      /* NB the query in constant memory is MAXDIM not MAXDIM_GPU 
         since constant memory larger than shared memory.
         This means we need to reformat the matrices into the larger 
         size if they are in the smaller class */
      for (int i = 0; i < orders[qdbi]; i++)
      {
        for (int j = i + 1; j < orders[qdbi]; j++)
        {
          char tabcode = (tableaux+qdbi*MAXDIM_GPU*MAXDIM_GPU)[INDEX2D(i,j,MAXDIM_GPU,MAXDIM_GPU)];
          qtab[INDEX2D(i,j,MAXDIM,MAXDIM)] = tabcode;
          qtab[INDEX2D(j,i,MAXDIM,MAXDIM)] = tabcode;
          float dist = (distmatrices+qdbi*MAXDIM_GPU*MAXDIM_GPU)[INDEX2D(i,j,MAXDIM_GPU,MAXDIM_GPU)];
          qdmat[INDEX2D(i,j,MAXDIM,MAXDIM)] = dist;
          qdmat[INDEX2D(j,i,MAXDIM,MAXDIM)] = dist;
        }
      }
      cutilSafeCall( hipMemcpyToSymbol(HIP_SYMBOL(c_qtab_symbol), qtab, MAXDIM*MAXDIM*sizeof(char)) );
      cutilSafeCall( hipMemcpyToSymbol(HIP_SYMBOL(c_qdmat_symbol), qdmat, MAXDIM*MAXDIM*sizeof(float)) );
      
      cutilSafeCall( hipMemcpyToSymbol(HIP_SYMBOL(c_qssetypes_symbol), qssetypes, MAXDIM*sizeof(char)) );
    }
  }
  else // single query mode - copy to constant memory
  {
    cutilSafeCall( hipMemcpyToSymbol(HIP_SYMBOL(c_qn_symbol), &qn, sizeof(qn)) );
    cutilSafeCall( hipMemcpyToSymbol(HIP_SYMBOL(c_qtab_symbol), qtab, MAXDIM*MAXDIM*sizeof(char)) );
    cutilSafeCall( hipMemcpyToSymbol(HIP_SYMBOL(c_qdmat_symbol), qdmat, MAXDIM*MAXDIM*sizeof(float)) );
    cutilSafeCall( hipMemcpyToSymbol(HIP_SYMBOL(c_qssetypes_symbol), qssetypes, MAXDIM*sizeof(char)) );
  }
  cutilCheckError( cutStopTimer(hTimer) );
  float qtime = cutGetTimerValue(hTimer);
  fprintf(stderr, "Copying query to constant memory (%s) took %f ms\n", 
          c_qtab_symbol,
          qtime);
}


static void usage(const char *progname)
{
  fprintf(stderr, "Usage: %s [-c] [-q dbfile]\n", progname);
  fprintf(stderr, "  -c : run on host CPU not GPU card\n");
  fprintf(stderr, "  -q dbfile : database is read from dbfile, list of query\n"
          "              ids is read from stdin\n");
  fprintf(stderr, "   -r restarts : number of restarts. Default %d\n",
          DEFAULT_MAXSTART);
  exit(1);
}


int main(int argc, char *argv[])
{
  CUTThread threadID[MAX_THREADS];
  int num_threads = 0;
  int exit_status = 0;
  char buf[MAX_LINE_LEN];
  char qtab[MAXDIM*MAXDIM];
  float qdmat[MAXDIM*MAXDIM];
  int qn;
  char qid[LABELSIZE+1];
  int ltype=0,lorder=0,lsoln=0;
  char cltype,clorder,clsoln;
  FILE *dbfp;
  unsigned int hTimer;
  int total_dbsize, large_dbsize, gpu_dbsize;
  double dbtime,runtime;
  hipPitchedPtr d_tableaux;
  hipPitchedPtr d_distmatrices;
  int *d_orders;
  int *scores = NULL;
  int *ssemaps = NULL;
  int *d_scores;
  int *d_ssemaps;
  hipError_t cuda_errcode;
  int i,j;
  char qssetypes[MAXDIM];
  int c;
  char *queryptr = NULL;
  int num_queries = 0;
  int large_query_count = 0;

  while ((c = getopt(argc, argv, "cq:r:")) != -1)
  {
    switch (c)
    {
      case 'c':
        use_gpu = false;
        break;

      case 'q':
        querydbmode = true;
        strncpy(dbfile, optarg, sizeof(dbfile)-1);
        break;

      case 'r':
        maxstart = atoi(optarg);
        break;
        
      default:
        usage(argv[0]);
        break;
    }
  }

  if (querydbmode)
  {
    cltype = 'T'; ltype = 1;
    clorder = 'T'; lorder = 1;
    clsoln = 'F'; lsoln = 0;
    if (!(queryid_list = (char *)malloc(LABELSIZE+1)))
    {
      fprintf(stderr, "malloc queryid_list failed\n");
      exit(1);
    }
    queryptr = queryid_list;
    while (!feof(stdin))
    {
      if (num_queries > 0)
      {
        if ((!(queryid_list = (char *)realloc(queryid_list, (num_queries+1)*(LABELSIZE+1)))))
        {
          fprintf(stderr, "realloc queryid_list failed\n");
          exit(1);
        }
      }
      if (!fgets(buf, MAX_LINE_LEN, stdin))
        break;
      strncpy(queryptr, buf, LABELSIZE);
      queryptr[LABELSIZE-1] = '\0';
      if (queryptr[strlen(queryptr)-1] == '\n')
        queryptr[strlen(queryptr)-1] = '\0';
      queryptr += (LABELSIZE+1);
      num_queries++;
    }
  }
  else
  {
    if (fscanf(stdin, "%s\n", dbfile) != 1)
    {
      fprintf(stderr, "ERROR reading dbfilename from stdin\n");
      exit(1);
    }
    if (fscanf(stdin, "%c %c %c\n", &cltype, &clorder, &clsoln) != 3)
    {
      fprintf(stderr, "ERROR reading options from stdin\n");
      exit(1);
    }
    if (cltype == 'T')
      ltype = 1;
    if (clorder == 'T')
      lorder = 1;
    if (clsoln == 'T')
      lsoln = 1;
    
    if (fscanf(stdin, "%8s %d\n", qid, &qn) != 2)
    {
      fprintf(stderr, "ERROR parsing query tableau header from stdin\n");
      exit(1);
    }
    if (parse_tableau(stdin, MAXDIM, qn, qtab) < 0)
    {
      fprintf(stderr, "ERROR parsing query tableau from stdin\n");
      exit(1);
    }
    if (parse_distmatrix(stdin, MAXDIM, qn, qdmat, 0) < 0)
    {
      fprintf(stderr, "ERROR parsing query distance matrix from stdin\n");
      exit(1);
    }
  }

  if (!ltype)
  {
    fprintf(stderr, "WARNING: LTYPE is always set to T\n");
    ltype = 1; cltype = 'T';
  }
  
  if (!(dbfp = fopen(dbfile, "r")))
  {
    fprintf(stderr, "ERROR opening db file %s\n", dbfile);
    exit(1);
  }

  fprintf(stderr, "Loading database...\n");
  cutilCheckError( cutCreateTimer(&hTimer) );
  cutilCheckError( cutResetTimer(hTimer) );
  cutilCheckError( cutStartTimer(hTimer) );
  total_dbsize = read_database(dbfp, &tableaux, &distmatrices, 
                               &large_tableaux, &large_distmatrices,
                               &orders, &names,
                               &large_orders, &large_names,
                               &large_dbsize);
  if (total_dbsize < 0)
  {
    fprintf(stderr, "ERROR loading database\n");
    exit(1);
  }
  gpu_dbsize = total_dbsize - large_dbsize;
  cutilCheckError( cutStopTimer(hTimer) );
  dbtime = cutGetTimerValue(hTimer);
  fprintf(stderr, "Loaded %d db entries (%d order > %d) in %f ms\n", 
          total_dbsize, large_dbsize, MAXDIM_GPU, dbtime);
          

  if (querydbmode)
  {
    /* Convert the list of query sids to list of indices in db for later
       rapid lookup.
       TODO: we should build a hash table rather than this highly 
       inefficient linear search for each query id, but it's only
       done once and db not that big... 
    */
    fprintf(stderr, "Building query index list...\n");
    cutilCheckError( cutResetTimer(hTimer) );
    cutilCheckError( cutStartTimer(hTimer) );
    if (!(query_dbindex_list = (dbIndex_t *)malloc(num_queries*sizeof(dbIndex_t))))
    {
      fprintf(stderr, "malloc query_dbindex_list failed\n");
      exit(1);
    }
    for (i = 0; i < num_queries; i++)
    {
/*      fprintf(stderr, "zzz %s\n", queryid_list+i*(LABELSIZE+1)); */
      bool found = false;
      for (j = 0; j < gpu_dbsize; j++) /* search 'small' structure dbase */
      {
        if (!strcasecmp(queryid_list+i*(LABELSIZE+1),names+j*(LABELSIZE+1)))
        {
          query_dbindex_list[i].large = false;
          query_dbindex_list[i].index = j;
          found = true;
          break;
        }
      }
      if (!found)
      {
        for (j = 0; j < large_dbsize; j++) /* search 'large' structure dbase*/
        {
          if (!strcasecmp(queryid_list + i*(LABELSIZE+1),
                           large_names + j*(LABELSIZE+1)))
          {
            query_dbindex_list[i].large = true;
            query_dbindex_list[i].index = j;
            large_query_count++;
            found = true;
            break;
          }
        }
      }
      if (!found)
      {
        fprintf(stderr, "ERROR: query %s not found\n", queryid_list+i*(LABELSIZE+1));
        exit(1);
      }
    }
    cutilCheckError( cutStopTimer(hTimer) );
    fprintf(stderr, "Built query index (%d queries (%d large)) in %f ms\n",
            num_queries, large_query_count, cutGetTimerValue(hTimer));
  }
  else
  {
    num_queries = 0; 
    query_dbindex_list = NULL;
    // set the qssetypes vector as main diagonal of the query tableau
    for (i = 0; i < qn; i++)
      qssetypes[i] = qtab[INDEX2D(i,i,MAXDIM,MAXDIM)];
  }
    
  /* TODO allow multiple GPUs (need one thread for each) */

  if (use_gpu)
  {
/*
    int devnum = cutGetMaxGflopsDeviceId();
    fprintf(stderr, "using max gflops device %d: ", devnum);
*/
    /* If there is a compute capability 2 device ("Fermi"
       architecture) (or higher) then use that, and do NOT use shared
       memory as it is faster to just rely on the new "NVIDIA Parallel
       DataCache (TM)" -- just use global memory for all (small and large)
       structures
    */

    int devnum, deviceCount, gflops,max_gflops=0, sel_devnum;
    hipGetDeviceCount(&deviceCount);
    if (deviceCount == 0)
    {
      fprintf(stderr, "There is no device supporting CUDA.\n");
      exit(1);
    }
    fprintf(stderr, "found %d CUDA devices\n", deviceCount);
    for (devnum = 0; devnum < deviceCount; devnum++)
    {  
      hipDeviceProp_t deviceProp;
      hipGetDeviceProperties(&deviceProp, devnum);
      if (deviceProp.major >= 2)
      {
        fprintf(stderr,
          "found Fermi architecture (compute capability %d.%d) device %d: %s\n",
                deviceProp.major, deviceProp.minor, devnum, deviceProp.name);
        sel_devnum = devnum;
        use_shared_memory = false;
        break;
      }
      else
      {
        gflops = deviceProp.multiProcessorCount * deviceProp.clockRate;
        fprintf(stderr, "device %d: %s\n", devnum,
                deviceProp.name);
        if (gflops > max_gflops)
        {
          max_gflops = gflops;
          sel_devnum = devnum;
          use_shared_memory = true;
        }
      }
    }
    
    fprintf(stderr, "using device %d: ", sel_devnum);
    hipDeviceProp_t deviceProp;
    hipGetDeviceProperties(&deviceProp, sel_devnum);
    fprintf(stderr, "%s\n", deviceProp.name);
    hipSetDevice( sel_devnum );
  }

  InitRandomNumberGenerator();

  fprintf(stderr, "maxstart = %d\n", maxstart);

  if (use_gpu)
  {
    /* setup execution configuration parameters */
    /* TODO optimize for different architectures (automatically) */
    dim3 dimGrid(128);          // blocks
    dim3 dimBlock(128);         // threads per block


    fprintf(stderr, "Execution configuration: Grid = (%d,%d,%d) Block = (%d,%d,%d)\n", dimGrid.x,dimGrid.y,dimGrid.z, dimBlock.x,dimBlock.y,dimBlock.z);

    if (dimBlock.x * dimGrid.x > MT_RNG_COUNT)
    {
      fprintf(stderr, "ERROR: can only have a max of %d threads in configuration\n", MT_RNG_COUNT);
      exit(1);
    }

    fprintf(stderr, "using shared memory for small db structs: %s\n",
            use_shared_memory ? "YES" : "NO");

    /* first do the 'small' db structures on GPU (with shared memory) */

    fprintf(stderr, "Copying database to device...\n");
    cutilCheckError( cutResetTimer(hTimer) );
    cutilCheckError( cutStartTimer(hTimer) );

    hipExtent tableaux_extent = make_hipExtent(MAXDIM_GPU, MAXDIM_GPU,
                                                 gpu_dbsize);
    cutilSafeCall( hipMalloc3D(&d_tableaux, tableaux_extent) );
    fprintf(stderr, "d_tableaux.pitch == %u xsize == %u ysize == %u\n", d_tableaux.pitch, d_tableaux.xsize, d_tableaux.ysize);

    hipExtent distmatrices_extent = make_hipExtent(MAXDIM_GPU*sizeof(float), MAXDIM_GPU, gpu_dbsize);
    cutilSafeCall( hipMalloc3D(&d_distmatrices, distmatrices_extent) );
    fprintf(stderr, "d_distmatrices.pitch == %u xsize == %u ysize == %u\n", d_distmatrices.pitch, d_distmatrices.xsize, d_distmatrices.ysize);

    cutilSafeCall( hipMalloc((void **)&d_orders, gpu_dbsize*sizeof(int)) );


    hipMemcpy3DParms copyParams = { 0 };
    // srcPtr is tricky: need to give pitch of row, #elements in row,
    // then height, omitting 3rd dimension (doesn't seem to be documented)
    // (I found this info on 28/1/2010 at 
    // http://sites.google.com/site/cudaiap2009/cookbook-1).
    // Note pitch of row on host is just MAXDIM_GPU, we don't need padding here
    copyParams.srcPtr = make_hipPitchedPtr((void*)tableaux, MAXDIM_GPU, MAXDIM_GPU, MAXDIM_GPU);

    fprintf(stderr, "srcPtr.pitch == %u\n", copyParams.srcPtr.pitch);

    copyParams.dstPtr = d_tableaux;
    copyParams.extent = tableaux_extent;
    copyParams.kind = hipMemcpyHostToDevice;
    cutilSafeCall( hipMemcpy3D(&copyParams) );

    
    hipMemcpy3DParms copyParams2 = { 0 };
    copyParams2.srcPtr = make_hipPitchedPtr((void*)distmatrices,
                                             MAXDIM_GPU*sizeof(float),
                                             MAXDIM_GPU, MAXDIM_GPU);
    fprintf(stderr, "distmatrices srcPtr.pitch == %u\n", copyParams2.srcPtr.pitch);
    copyParams2.dstPtr = d_distmatrices;
    copyParams2.extent = distmatrices_extent;
    copyParams2.kind = hipMemcpyHostToDevice;
    cutilSafeCall( hipMemcpy3D(&copyParams2) );

    cutilSafeCall( hipMemcpy(d_orders, orders, gpu_dbsize*sizeof(int),
                              hipMemcpyHostToDevice) );

    cutilCheckError( cutStopTimer(hTimer) );
    dbtime = cutGetTimerValue(hTimer);
    fprintf(stderr, "Copied %d entries to GPU in %f ms\n", gpu_dbsize, dbtime);

    /* allocate space for output */
    cutilSafeCall( hipMalloc((void **)&d_scores, gpu_dbsize*sizeof(int)));
    if (!(scores = (int *)malloc(gpu_dbsize*sizeof(int))))
    {
      fprintf(stderr, "malloc scores failed\n");
      goto bye;
    }
    if (lsoln)
    {
      cutilSafeCall( hipMalloc((void **)&d_ssemaps, gpu_dbsize*MAXDIM*sizeof(int)));
      if (!(ssemaps = (int *)malloc(gpu_dbsize*MAXDIM*sizeof(int))))
      {
        fprintf(stderr, "malloc ssemaps failed\n");
        goto bye;
      }
    }
      
    int query_count = (num_queries == 0 ? 1 : num_queries);
    for (int qi = 0; qi < query_count; qi++)
    {
      if (use_shared_memory)
        copyQueryToConstantMemory(qi, qn, qtab, qdmat, qssetypes, qid,
                                  "c_qn", "c_qtab", "c_qdmat", "c_qssetypes");
      else
        copyQueryToConstantMemory(qi, qn, qtab, qdmat, qssetypes, qid,
                                  "c_qn_noshared_small",
                                  "c_qtab_noshared_small", 
                                  "c_qdmat_noshared_small", 
                                  "c_qssetypes_noshared_small");
        

      printf("# cudaSaTabsearch LTYPE = %c LORDER = %c LSOLN = %c\n",
             cltype, clorder, clsoln);
      printf("# QUERY ID = %-8s\n", qid);
      printf("# DBFILE = %-80s\n", dbfile);


      /* launch thread to do large db structs on host */
      searchParams_t host_params;
      host_params.ltype = ltype;
      host_params.lorder = lorder;
      host_params.lsoln = lsoln;
      host_params.maxstart = maxstart;
      host_params.num_queries = num_queries;
      host_params.query_dbindex_list = query_dbindex_list;
      host_params.single_query_qid = qi; 
      memcpy(host_params.qtab, qtab, sizeof(qtab));
      memcpy(host_params.qdmat, qdmat, sizeof(qdmat));
      memcpy(host_params.qid, qid, sizeof(qid));
      host_params.qn = qn;
      host_params.qssetypes = qssetypes;
      host_params.maxdim = MAXDIM;
      host_params.dbsize = large_dbsize;
      host_params.tableaux = large_tableaux;
      host_params.distmatrices = large_distmatrices;
      host_params.orders = large_orders;
      host_params.names = large_names;

//XXX      threadID[num_threads++] = cutStartThread((CUT_THREADROUTINE)tabsearch_host_thread, &host_params);


      fprintf(stderr, "Executing simulated annealing tableaux match kernel (%sshared memory) on GPU for qid %s...\n", use_shared_memory ? " " : "no ", qid);
      cutilSafeCall( hipDeviceSynchronize() );
      cutilCheckError( cutResetTimer(hTimer) );
      cutilCheckError( cutStartTimer(hTimer) );
      if (use_shared_memory)
        sa_tabsearch_gpu<<<dimGrid,dimBlock>>>(gpu_dbsize,
                                               lorder, 
                                               lsoln,
                                               maxstart,
                                               d_tableaux, tableaux_extent,
                                               d_orders,
                                               d_distmatrices, distmatrices_extent,
                                               d_scores,
                                               d_ssemaps);
      else
        sa_tabsearch_gpu_noshared_small<<<dimGrid,dimBlock>>>(gpu_dbsize,
                                               lorder, 
                                               lsoln,
                                               maxstart,
                                               d_tableaux, tableaux_extent,
                                               d_orders,
                                               d_distmatrices, distmatrices_extent,
                                               d_scores,
                                               d_ssemaps);

      cuda_errcode = hipGetLastError();
      if (cuda_errcode != hipSuccess)
      {
        fprintf(stderr, "kernel launch failed: %s\n", hipGetErrorString(cuda_errcode));
        exit_status = 1;
        goto bye;
      }

      cutilSafeCall( hipDeviceSynchronize() );
      cutilCheckError( cutStopTimer(hTimer) );
      runtime = cutGetTimerValue(hTimer);
      fprintf(stderr,  "GPU execution time %f ms\n", runtime);
      fprintf(stderr,  "%f million iterations/sec\n", ((float)gpu_dbsize * ((float)maxstart * (float)MAXITER) / (runtime/1000)) / 1.0e6);
      
      /* Get results from device */
      cutilSafeCall( hipMemcpy(scores, d_scores, gpu_dbsize*sizeof(int), 
                                hipMemcpyDeviceToHost) );
      if (lsoln)
        cutilSafeCall( hipMemcpy(ssemaps, d_ssemaps, 
                                  gpu_dbsize*MAXDIM*sizeof(int),
                                  hipMemcpyDeviceToHost) );

      /* Wait for host thread */
//XXX      cutWaitForThreads(threadID, num_threads);
//XXX      --num_threads;



      /* TODO we could reduce wasted time waiting by running all host 
         (large db) queries in the one thread instead of matching up
         with GPU query in this loop (actuall, more like the other way
         around usually, the GPU ends up idle while host is still runnign since
         the latter is so much slower even though it has very few
         db entries unlike GPU) */

      for (i = 0; i < gpu_dbsize; i++)
      {
        printf("%-8s  %d\n", names+i*(LABELSIZE+1), scores[i]);
        if (lsoln)
          for (int k = 0; k < qn; k++)
            if (ssemaps[i*MAXDIM + k] >= 0)
              printf("%3d %3d\n", k+1, ssemaps[i*MAXDIM + k]+1);
      }
    }

    cutilSafeCall( hipFree(d_tableaux.ptr) );
    cutilSafeCall( hipFree(d_distmatrices.ptr) );
    cutilSafeCall( hipFree(d_orders) );
    cutilSafeCall( hipFree(d_scores) );
    free(scores); scores = NULL;
    if (lsoln)
    {
      cutilSafeCall( hipFree(d_ssemaps) );
      free(ssemaps); ssemaps = NULL;
    }

    /* now do the 'large' db structures on GPU (not using shared memory) */
    if (large_dbsize > 0)
    {
      fprintf(stderr, "Copying large structure database to device...\n");
      cutilCheckError( cutResetTimer(hTimer) );
      cutilCheckError( cutStartTimer(hTimer) );

      tableaux_extent = make_hipExtent(MAXDIM, MAXDIM, large_dbsize);
      cutilSafeCall( hipMalloc3D(&d_tableaux, tableaux_extent) );
      fprintf(stderr, "d_tableaux.pitch == %u xsize == %u ysize == %u\n", d_tableaux.pitch, d_tableaux.xsize, d_tableaux.ysize);

      distmatrices_extent = make_hipExtent(MAXDIM*sizeof(float), MAXDIM, large_dbsize);
      cutilSafeCall( hipMalloc3D(&d_distmatrices, distmatrices_extent) );
      fprintf(stderr, "d_distmatrices.pitch == %u xsize == %u ysize == %u\n", d_distmatrices.pitch, d_distmatrices.xsize, d_distmatrices.ysize);

      cutilSafeCall( hipMalloc((void **)&d_orders, large_dbsize*sizeof(int)) );


      hipMemcpy3DParms copyParamsl = { 0 };
      // srcPtr is tricky: need to give pitch of row, #elements in row,
      // then height, omitting 3rd dimension (doesn't seem to be documented)
      // (I found this info on 28/1/2010 at 
      // http://sites.google.com/site/cudaiap2009/cookbook-1).
      // Note pitch of row on host is just MAXDIM_GPU, we don't need padding here
      copyParamsl.srcPtr = make_hipPitchedPtr((void*)large_tableaux, MAXDIM, MAXDIM, MAXDIM);

      fprintf(stderr, "srcPtr.pitch == %u\n", copyParamsl.srcPtr.pitch);

      copyParamsl.dstPtr = d_tableaux;
      copyParamsl.extent = tableaux_extent;
      copyParamsl.kind = hipMemcpyHostToDevice;
      cutilSafeCall( hipMemcpy3D(&copyParamsl) );


      hipMemcpy3DParms copyParams2l = { 0 };
      copyParams2l.srcPtr = make_hipPitchedPtr((void*)large_distmatrices,
                                                MAXDIM*sizeof(float),
                                                MAXDIM, MAXDIM);
      fprintf(stderr, "distmatrices srcPtr.pitch == %u\n", copyParams2l.srcPtr.pitch);
      copyParams2l.dstPtr = d_distmatrices;
      copyParams2l.extent = distmatrices_extent;
      copyParams2l.kind = hipMemcpyHostToDevice;
      cutilSafeCall( hipMemcpy3D(&copyParams2l) );

      cutilSafeCall( hipMemcpy(d_orders, large_orders, large_dbsize*sizeof(int),
                                hipMemcpyHostToDevice) );

      cutilCheckError( cutStopTimer(hTimer) );
      dbtime = cutGetTimerValue(hTimer);
      fprintf(stderr, "Copied %d large entries to GPU in %f ms\n", large_dbsize, dbtime);

      /* allocate space for output */
      cutilSafeCall( hipMalloc((void **)&d_scores, large_dbsize*sizeof(int)));
      if (!(scores = (int *)malloc(large_dbsize*sizeof(int))))
      {
        fprintf(stderr, "malloc scores failed\n");
        goto bye;
      }
      if (lsoln)
      {
        cutilSafeCall( hipMalloc((void **)&d_ssemaps, large_dbsize*MAXDIM*sizeof(int)));
        if (!(ssemaps = (int *)malloc(large_dbsize*MAXDIM*sizeof(int))))
        {
          fprintf(stderr, "malloc ssemaps failed\n");
          goto bye;
        }
      }

      for (int qi = 0; qi < query_count; qi++)
      {
        copyQueryToConstantMemory(qi, qn, qtab, qdmat, qssetypes, qid,
                                  "c_qn_noshared", "c_qtab_noshared", 
                                  "c_qdmat_noshared", "c_qssetypes_noshared");

        printf("# cudaSaTabsearch LTYPE = %c LORDER = %c LSOLN = %c\n",
               cltype, clorder, clsoln);
        printf("# QUERY ID = %-8s\n", qid);
        printf("# DBFILE = %-80s\n", dbfile);



        fprintf(stderr, "Executing simulated annealing tableaux match kernel (no shared memory) on GPU for qid %s...\n",qid);
        cutilSafeCall( hipDeviceSynchronize() );
        cutilCheckError( cutResetTimer(hTimer) );
        cutilCheckError( cutStartTimer(hTimer) );
        sa_tabsearch_gpu_noshared<<<dimGrid,dimBlock>>>(large_dbsize,
                                                        lorder, 
                                                        lsoln,
                                                        maxstart,
                                               d_tableaux, tableaux_extent,
                                               d_orders,
                                               d_distmatrices, distmatrices_extent,
                                               d_scores,
                                               d_ssemaps);
        cuda_errcode = hipGetLastError();
        if (cuda_errcode != hipSuccess)
        {
          fprintf(stderr, "kernel launch failed: %s\n", hipGetErrorString(cuda_errcode));
          exit_status = 1;
          goto bye;
        }

        cutilSafeCall( hipDeviceSynchronize() );
        cutilCheckError( cutStopTimer(hTimer) );
        runtime = cutGetTimerValue(hTimer);
        fprintf(stderr,  "GPU (no shared memory) execution time %f ms\n", runtime);
        fprintf(stderr,  "%f million iterations/sec\n", ((float)large_dbsize * ((float)maxstart * (float)MAXITER) / (runtime/1000)) / 1.0e6);

        /* Get results from device */
        cutilSafeCall( hipMemcpy(scores, d_scores, large_dbsize*sizeof(int), 
                                  hipMemcpyDeviceToHost) );
        if (lsoln)
          cutilSafeCall( hipMemcpy(ssemaps, d_ssemaps, 
                                    large_dbsize * MAXDIM * sizeof(int),
                                    hipMemcpyDeviceToHost) );

        for (i = 0; i < large_dbsize; i++)
        {
          printf("%-8s  %d\n", large_names+i*(LABELSIZE+1), scores[i]);
          if (lsoln)
            for (int k = 0; k < qn; k++)
              if (ssemaps[i*MAXDIM + k] >= 0)
                printf("%3d %3d\n", k+1, ssemaps[i*MAXDIM + k]+1);
        }
      }
    }
  }
  else
  {
    /* running on host CPU */

    searchParams_t host_params;
    host_params.ltype = ltype;
    host_params.lorder = lorder;
    host_params.lsoln = lsoln;
    host_params.maxstart = maxstart;
    host_params.num_queries = num_queries;
    host_params.single_query_qid = -1;
    host_params.query_dbindex_list = query_dbindex_list;

    memcpy(host_params.qtab, qtab, sizeof(qtab));
    memcpy(host_params.qdmat, qdmat, sizeof(qdmat));
    memcpy(host_params.qid, qid, sizeof(qid));
    host_params.qn = qn;
    host_params.qssetypes = qssetypes;
    host_params.maxdim = MAXDIM_GPU;
    host_params.dbsize = gpu_dbsize;

    /* first do small structure db */
    host_params.tableaux = tableaux;
    host_params.distmatrices = distmatrices;
    host_params.orders = orders;
    host_params.names = names;
    
    tabsearch_host_thread(&host_params);

    /* then large structure db */
    if (large_dbsize > 0)
    {
      host_params.maxdim = MAXDIM;
      host_params.dbsize = large_dbsize;
      host_params.tableaux = large_tableaux;
      host_params.distmatrices = large_distmatrices;
      host_params.orders = large_orders;
      host_params.names = large_names;
      tabsearch_host_thread(&host_params);
    }
  }

bye:
  /* cleanup and exit */
  free(tableaux);
  free(distmatrices);
  free(orders);
  free(names);
  free(scores);
  free(large_tableaux);
  free(large_distmatrices);
  free(large_names);
  free(large_orders);
  if (lsoln)
    free(ssemaps);
  cutilCheckError( cutDeleteTimer( hTimer) );
  if (use_gpu)
  {
    if (large_dbsize > 0)
    {
      cutilSafeCall( hipFree(d_tableaux.ptr) );
      cutilSafeCall( hipFree(d_distmatrices.ptr) );
      cutilSafeCall( hipFree(d_orders) );
      cutilSafeCall( hipFree(d_scores) );
      if (lsoln)
        cutilSafeCall( hipFree(d_ssemaps) );
    }
    hipDeviceReset();
  }
  exit(exit_status);
}

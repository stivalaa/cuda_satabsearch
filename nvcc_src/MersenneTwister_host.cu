#include "hip/hip_runtime.h"

/*****************************************************************************
 *
 * This file contains code adapted from CUDA SDK 2.3 
 *
 * The Mersenne Twister RNG kernel from the CUDA SDK, modified so that
 * instead of standalone generating numbers and storing them,
 * it is called to get next number like the standard C library rand() etc,.
 * from the device only (not a __global__ kernel any more).
 *
 * $Id: MersenneTwister_host.cu 3350 2010-02-18 00:32:08Z alexs $
 *****************************************************************************/

/*
 * Copyright 1993-2009 NVIDIA Corporation.  All rights reserved.
 *
 * NVIDIA Corporation and its licensors retain all intellectual property and 
 * proprietary rights in and to this software and related documentation and 
 * any modifications thereto.  Any use, reproduction, disclosure, or distribution 
 * of this software and related documentation without an express license 
 * agreement from NVIDIA Corporation is strictly prohibited.
 * 
 */

#include <stdio.h>
#include <cutil_inline.h>
#include "MersenneTwister.h"


/* __device__ static mt_struct_stripped ds_MT[MT_RNG_COUNT]; */
static mt_struct_stripped h_MT[MT_RNG_COUNT];



//Load twister configurations
void loadMTGPU(const char *fname){
    FILE *fd = fopen(fname, "rb");
    if(!fd){
        printf("initMTGPU(): failed to open %s\n", fname);
        printf("TEST FAILED\n");
        exit(0);
    }
    if( !fread(h_MT, sizeof(h_MT), 1, fd) ){
        printf("initMTGPU(): failed to load %s\n", fname);
        printf("TEST FAILED\n");
        exit(0);
    }
    fclose(fd);
}

//Initialize/seed twister for current GPU context
void seedMTGPU(unsigned int seed){
    int i;
    //Need to be thread-safe
    mt_struct_stripped *MT = (mt_struct_stripped *)malloc(MT_RNG_COUNT * sizeof(mt_struct_stripped));

    for(i = 0; i < MT_RNG_COUNT; i++){
        MT[i]      = h_MT[i];
        MT[i].seed = seed;
    }
    CUDA_SAFE_CALL( hipMemcpyToSymbol(HIP_SYMBOL("ds_MT"), MT, sizeof(h_MT)) );

    free(MT);
}

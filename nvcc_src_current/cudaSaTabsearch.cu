#include "hip/hip_runtime.h"
/*****************************************************************************
 * 
 * File:    cudaSaTabsearch.cu
 * Author:  Alex Stivala
 * Created: January 2010
 *
 * CUDA host code for simulated annealing tableau matching (discrete).
 * This is a CUDA implemenation of the FORTRAN subroutine TSAMTD.
 * Since the GPU has limited memory (and specifically, very limited
 * per block shared memory), we split the database into 'small' and
 * 'large' structures. The small ones can run on the GPU in shared memory,
 * the large ones cannot so we either have to not use shared memory
 * (OK, but a bit slower) or run them on the host.
 * When runnign on the host, we can simultaneously run the GPU and
 * host in separate threads. For multiple GPU cards, CUDA also requires
 * that there is a separate host thread for each GPU, so this program
 * is multithreaded: each thread is either for a separate GPU or for
 * running the same kernel (but compiled for host) on the host CPU.
 *
 * Usage: cudaSaTabsearch [-c] [-q dbfile] [-r restarts] < inputfile
 *
 * -c : run on host CPU not GPU card
 *
 * -q : query list mode: instead of reading query data on stdin
 *      just as in the original Fortran version tlocsd, a list
 *      of query sids to be read from the database is read on stdin (one per
 *      line),
 *      and db filenaame is specified on command
 *      line. In this mode options are assumed as LORDER=T, LTYPE=T,
 *      LSOLN=N. The output is still to stdout, but each query following
 *      immediately from the previous (can parse using the  header comment
 *      niformation lines as separators.
 *
 * -r restarts: number of restarts (iterations of cooling schedule).
 *              Should be a multiple of blocksize. Defaults to 128.
 *
 * The 'database' to search is an ASCII file of  tableaux
 * (Omega matrices) in format described in rdtabd.f.
 *
 * The results are printed to stdout as 
 *
 * name rawscore norm2score z-score p-value
 *
 *
 * Both the name of the database file to read, and the actual
 * query tableau are read from stdin. 
 * The first line is the name
 * of the database file.
 * The second line is for options. There are currently 3 logical
 * options, for SSE type constraint (only allow SSEs of same type ot
 * match) and ordering constraint (disallow out of sequence order 
 * matches). The third is to output not just the scores but also solution
 * vector values.
 * They are single character logical values (T or F).
 * First is type, second is order, third is solution output,
 * separated by one space.
 *
 * The subsequent lines are multiple query structures (tableau and
 * distance matrix for each), separated by a blank line. I.e. the
 * same format as the database.
 *
 * The tableau is in the same format as
 * each tableau entry in the database i.e.:
 *
 * The first line of an entry is the identifier and
 * order of tableau (i.e. dimension of square array), then
 * each subsequent row is a row of the tableau, lower triangle
 * only (since it is symmetric).
 * The diagonal entries are meaningless (self-angle) in tableaux,
 * and are included instead to specify the SSE type, with
 * the following codes:
 *
 * e     beta strand
 * xa    alpha helix
 * xi    pi helix
 * xg    3_10 helix
 *
 * Width of identifier is 8 chars, blank padded on right,
 * width of order is 4 digits, blank padded on left.
 * There is a single space between identifier and order.
 * Each entry in tableau is two characters, with a space betwen
 * each on a line, and one line
 * per row of matrix.
 *
 * Following the tableau is the distance matrix.
 * Each row is a row of the distance matrix, lower triangle
 * only (since it is symmetric).
 * The diagonal entries are meaningless (self-distance)
 * and are included instead to specify the SSE type, with
 * the following codes:
 * 
 * 0.000 beta strand
 * 1.000 alpha helix
 * 2.000 pi helix
 * 3.000 3_10 helix
 * 
 * Each entry in matrix is in Angstroms format
 * F6.3 with a space between each on a line, and one line
 * per row of matrix.
 * 
 * 
 * E.g.:
 * 
 * /local/charikar/astivala/tableauxdb/astral/tableauxdistmatrixdb.ascii
 *  T T F
 * D1UBIA_    8
 * e  
 * OT e  
 * LE RT xa 
 * PD OS RD xg 
 * RT LE RT LS e  
 * LE RD LE LS OT e  
 * RT LS LS RD PE OS xg 
 * PE RT LE RD OT PE RT e  
 *  0.000 
 *  4.501  0.000 
 *  1.662 10.386  1.000 
 * 16.932 17.644  9.779  3.000 
 * 10.588 13.738 11.815 10.527  0.000 
 * 15.025 18.692 17.143 15.341  6.466  0.000 
 * 15.298 17.276 16.276 20.075 13.264 11.610  3.000 
 *  7.549 11.072 12.248 12.446  4.583  9.903 15.689  0.000 
 *
 * D1AE6H1   13
 * e  
 * PD e  
 * OT OS e  
 * LS LS RD xg 
 * LE LE RT RT e  
 * RT RT LE LE OT e  
 * RT RT LE LE OT PE e  
 * LE LE RT RT PE OT PE e  
 * RT OT PE LS OT LE PE OT e  
 * PE PE OT LS PE RT OT PE OT e  
 * OT OT PE RD OS LE PE OT PE OT xg 
 * OT RT PE RD OT PE PE OT PE OT PD e  
 * PD PE OT LS LE RT RT LE OT PE OT OT e  
 *  0.000 
 * 19.130  0.000 
 *  8.850 13.371  0.000 
 * 14.608 29.221 15.945  3.000 
 * 12.469 19.135 11.231 16.008  0.000 
 * 18.479 21.128 16.959 21.982  6.730  0.000 
 * 16.153 22.704 13.140 13.210  6.909 10.709  0.000 
 * 20.850 24.610 16.558 16.527 10.946 12.552  4.935  0.000 
 * 15.604 18.394  8.791 14.366 11.402 16.188  8.316  9.609  0.000 
 * 13.949 13.565  5.751 17.301 10.771 15.314 10.725 12.661  4.876  0.000 
 * 24.234 12.620 19.140 31.786 17.166 14.790 20.733 21.202 20.224 16.665  3.000 
 *  9.731 17.355  9.936 16.942  3.841  8.797 10.327 14.566 13.021 11.226 17.023  0.000 
 * 16.856  5.985 12.706 27.454 15.011 16.146 19.829 22.156 17.744 13.079  9.541 12.996  0.000 
 *
 *****************************************************************************/

#define CUDASATABSEARCH_MAIN 1

#include <stdlib.h>
#include <stdio.h>
#include <getopt.h>
#include <time.h>
#include <string.h>
#include <multithreading.h>
#include <hip/hip_runtime_api.h>
#include <helper_timer.h>
#include <hiprand/hiprand_kernel.h>
#include "parsetableaux.h"
#include "cudaSaTabsearch_kernel.h"
#include "cudaGetDeviceConstantAddresses.h"
#include "gumbelstats.h"



/*****************************************************************************
 *
 * Type definitions
 *
 *****************************************************************************/

/* dbIndex_t is for the query list mode, an array of these gives for each
   query the index in the appropriate ('small' or 'large' according to the
   large flag) tableaux and distmatrix db arrays */
typedef struct dbIndex_s 
{
    bool large;  /* true if query is 'large' (>MAXDIM_GPU) structure */
    int  index;  /* index in tableaux and distmatrix db list, or 'large'
                    tableaux and distmatrix db list if large is true */
} dbIndex_t;

/* queryData_t is a struct containing the data for a single query structure */
typedef struct queryData_s {
    char qtab[MAXDIM*MAXDIM];     /* the query tableau */
    float qdmat[MAXDIM*MAXDIM];   /* the query distmatrix*/
    char qid[LABELSIZE+1];        /* the query identifier*/
    int qn;                       /* the query order */
    char *qssetypes;              /* the query SSE types*/
} queryData_t;

/* searchParams_t is a struct for parameter to tableau search functions
   dcelared as CUT_THREADROUTINE to be callable as threads */
typedef struct searchParams_s
{
    int ltype; int lorder; int lsoln; /* type,order,soln flags */
    unsigned long maxstart;           /* number of restarts */
    unsigned long maxdim;             /*dimension of tableaux, distmatrices here */
    int num_queries;        /* number of queries */
    int single_query_qid; /* if >=0, do only the one at this index */
    dbIndex_t *query_dbindex_list; /* the query db index (query list mode) */
                                   /* OR query data (notq query list mode) : */
    char  *q_tableaux;                     /* query tableaux */
    float *q_distmatrices;                 /* query distance matrices */
    int   *q_orders;                       /* sizes of query tableaux */
    char  *q_names;                        /* names of queries */

    unsigned long dbsize;   /* number of entries in the db */
    char *tableaux;         /* the tableaux database */
    float *distmatrices;    /* the distance matrices database */
    int   *orders;          /* orders of entries in db */
    char  *names;           /* names of entries in db */
    
} searchParams_t;


/*****************************************************************************
 *
 * Globals
 *
 *****************************************************************************/

static char dbfile[MAX_LINE_LEN];   /* database file name */
static bool use_gpu = true;   /* use the GPU */
static bool use_shared_memory = true; /* use GPU shared mem for db structs */
static char *tableaux, *large_tableaux; /* small and large tableaux */
static float *distmatrices, *large_distmatrices; /* same for dist.matrices*/
static int *orders, *large_orders; /* and for orders */
static char *names, *large_names;  /* and names */
static bool querydbmode = false;   /* use list of query ids in db */
static char *queryid_list = NULL;  /* this is the list of query ids */
static dbIndex_t *query_dbindex_list = NULL; /* and their indices in db */

static int maxstart = DEFAULT_MAXSTART; /* number of restarts */

static char *query_tableaux;       /* query tableaux */
static float *query_distmatrices;; /* query dist.matrices*/
static int *query_orders;          /* query orders */
static char *query_names;          /* query names */



/*
 * init_rng()
 *
 * Initialize CURAND pseudrandom number generator
 * See CUDA Toolkit 4.1 CURAND Guide (p.21)
 *
 * Parameters:
 *    state - CURAND state for random number generation
 *
 */
__global__ void init_rng(hiprandState *state)
{
  int tid=blockIdx.x*blockDim.x+threadIdx.x;

  /* give each therad same seed, different sequence number, no offset */
  hiprand_init(1234, tid, 0, &state[tid]);
}




///////////////////////////////////////////////////////////////////////////////
// Common host and device function 
///////////////////////////////////////////////////////////////////////////////
//ceil(a / b)
extern "C" int iDivUp(int a, int b){
    return ((a % b) != 0) ? (a / b + 1) : (a / b);
}

//floor(a / b)
extern "C" int iDivDown(int a, int b){
    return a / b;
}

//Align a to nearest higher multiple of b
extern "C" int iAlignUp(int a, int b){
    return ((a % b) != 0) ?  (a - a % b + b) : a;
}

//Align a to nearest lower multiple of b
extern "C" int iAlignDown(int a, int b){
    return a - a % b;
}






/*
 * tabsearch_host_thread - run the tableau search kernel on host CPU
 *
 * Started as a thread by cutStartThread in main
 *
 * Parameters:
 *   params - paramter block for thread. See comments on searchParams_t defn.
 *
 * Return value: None.
 *
 */
static CUT_THREADPROC tabsearch_host_thread(searchParams_t *params)
{
  /* extern declartions of host version of gpu constant memory */
  extern int c_qn_host;    // query structure size
  extern char c_qtab_host[MAXDIM*MAXDIM];  // query tableau
  extern float c_qdmat_host[MAXDIM*MAXDIM];  // query distance matrix
  extern char c_qssetypes_host[MAXDIM]; // main diagonal of c_qn


  StopWatchInterface *hTimer = NULL;
  double runtime;
  int *ssemaps;
  int i,j;
  char qid[LABELSIZE+1];
  int *scores;
  double norm2score,zscore,pvalue;
  char qssetypes[MAXDIM];

  int query_count = (params->query_dbindex_list && params->single_query_qid >= 0
                     ? 1 : params->num_queries);

  hipExtent tableaux_extent = {params->maxdim, params->maxdim,
                                params->dbsize};
  hipPitchedPtr tableaux_pp = {params->tableaux, params->maxdim,
                                params->maxdim, params->dbsize};
  hipExtent distmatrices_extent = {params->maxdim*sizeof(float), 
                                    params->maxdim,
                                    params->maxdim};
  hipPitchedPtr distmatrices_pp = {params->distmatrices, 
                                    params->maxdim*sizeof(float),
                                    params->maxdim,
                                    params->maxdim};


  /* allocate space for output */
  if (!(scores = (int *)malloc(params->dbsize*sizeof(int))))
  {
    fprintf(stderr, "malloc scores failed\n");
    return;
  }
  if (!(ssemaps = (int *)malloc(params->dbsize*MAXDIM*sizeof(int))))
  {
    fprintf(stderr, "malloc ssemaps failed\n");
    return;
  }

  for (int qi = 0; qi < query_count; qi++)
  {
    if (params->query_dbindex_list)
    {
      dbIndex_t *dbindex_entry =  params->single_query_qid >= 0 ? 
        &params->query_dbindex_list[params->single_query_qid] :
        &params->query_dbindex_list[qi];
      int qdbi = dbindex_entry->index;

      if (dbindex_entry->large) /* query in 'large' struct db */
      {
        strncpy(qid, large_names+qdbi*(LABELSIZE+1), LABELSIZE);
        c_qn_host = large_orders[qdbi];
        memcpy(c_qtab_host, large_tableaux+qdbi*MAXDIM*MAXDIM,
               MAXDIM*MAXDIM*sizeof(char));
        memcpy(c_qdmat_host, large_distmatrices+qdbi*MAXDIM*MAXDIM,
               MAXDIM*MAXDIM*sizeof(float));
        /* NB the query in constant memory is MAXDIM not MAXDIM_GPU 
           since constant memory larger than shared memory. */
        // set the qssetypes vector as main diagonal of the query tableau
        for (i = 0; i < c_qn_host; i++)
          c_qssetypes_host[i] = (large_tableaux+qdbi*MAXDIM*MAXDIM)[INDEX2D(i,i,MAXDIM,MAXDIM)];
      }
      else /* query in 'small' struct db */
      {
        strncpy(qid, names+qdbi*(LABELSIZE+1), LABELSIZE);
        c_qn_host = orders[qdbi];
        
        /* NB the query in constant memory is MAXDIM not MAXDIM_GPU 
           since constant memory larger than shared memory.
           This means we need to reformat the matrices into the larger 
             size if they are in the smaller class */
        for (i = 0; i < orders[qdbi]; i++)
        {
          for (j = i + 1; j < orders[qdbi]; j++)
          {
            char tabcode = (tableaux+qdbi*MAXDIM_GPU*MAXDIM_GPU)[INDEX2D(i,j,MAXDIM_GPU,MAXDIM_GPU)];
            c_qtab_host[INDEX2D(i,j,MAXDIM,MAXDIM)] = tabcode;
            c_qtab_host[INDEX2D(j,i,MAXDIM,MAXDIM)] = tabcode;
            float dist = (distmatrices+qdbi*MAXDIM_GPU*MAXDIM_GPU)[INDEX2D(i,j,MAXDIM_GPU,MAXDIM_GPU)];
            c_qdmat_host[INDEX2D(i,j,MAXDIM,MAXDIM)] = dist;
            c_qdmat_host[INDEX2D(j,i,MAXDIM,MAXDIM)] = dist;
          }
        }
        // set the qssetypes vector as main diagonal of the query tableau
        for (i = 0; i < c_qn_host; i++)
          c_qssetypes_host[i] = (tableaux+qdbi*MAXDIM_GPU*MAXDIM_GPU)[INDEX2D(i,i,MAXDIM_GPU,MAXDIM_GPU)];
      }
    }
    else
    {
      strncpy(qid, params->q_names+qi*(LABELSIZE+1), LABELSIZE);
      c_qn_host = params->q_orders[qi];
      memcpy(c_qtab_host, params->q_tableaux+qi*MAXDIM*MAXDIM, sizeof(c_qtab_host));
      memcpy(c_qdmat_host, params->q_distmatrices+qi*MAXDIM*MAXDIM, sizeof(c_qdmat_host));
      // set the qssetypes vector as main diagonal of the query tableau
      for (i = 0; i < params->q_orders[qi]; i++)
        qssetypes[i] = (params->q_tableaux+qi*MAXDIM*MAXDIM)[INDEX2D(i,i,MAXDIM,MAXDIM)];
      memcpy(c_qssetypes_host, qssetypes, sizeof(c_qssetypes_host));
    }
    
    printf("# cudaSaTabsearch LTYPE = %c LORDER = %c LSOLN = %c\n",
           params->ltype ? 'T' : 'F' , 
           params->lorder ? 'T' : 'F' , 
           params->lsoln ? 'T' : 'F');
    printf("# QUERY ID = %-8s\n", qid);
    printf("# DBFILE = %-80s\n", dbfile);
      
    fprintf(stderr, "Executing simulated annealing tableaux match kernel on host for query %s...\n", qid);
    sdkCreateTimer(&hTimer) ;
    sdkResetTimer(&hTimer) ;
    sdkStartTimer(&hTimer) ;
    int state = 0; /*unused*/
    sa_tabsearch_host(params->dbsize,
                      params->lorder, 
                      params->lsoln,
                      params->maxstart,
                      tableaux_pp, tableaux_extent,
                      params->orders,
                      distmatrices_pp, distmatrices_extent,
                      scores,
                      ssemaps,
                      &state);
    sdkStopTimer(&hTimer);
    runtime = sdkGetTimerValue(&hTimer);
    fprintf(stderr,  "host execution time %f ms\n", runtime);
    fprintf(stderr,  "%f million iterations/sec\n", (params->dbsize * (params->maxstart * MAXITER) / (runtime/1000)) / 1.0e6);
    
    for (i = 0; i < params->dbsize; i++)
    {
/*      printf("%-8s  %d\n", params->names+i*(LABELSIZE+1), scores[i]); */
      norm2score = norm2(scores[i], c_qn_host, params->orders[i]);
      zscore = z_gumbel(norm2score, gumbel_a, gumbel_b);
      pvalue = pv_gumbel(zscore);
      printf("%-8s %d %g %g %g\n", params->names+i*(LABELSIZE+1),
             scores[i], norm2score, zscore, pvalue);
      if (params->lsoln)
        for (int k = 0; k < c_qn_host; k++)
          if (ssemaps[i*MAXDIM + k] >= 0)
            printf("%3d %3d\n", k+1, ssemaps[i*MAXDIM + k]+1);
    }
  }
  free(scores);
  if (params->lsoln)
    free(ssemaps);
}



/*
 * copyQueryToConstantMemory() - copy the query data to device constant memory
 *
 *
 * Parameters:
 *   qi - the query index of the query to copy. 
 *        Otherwise (query_dbinex_list is NULL), these used:
 *   qn -query order
 *   qtab - query tableau  (in/out: may be set here)
 *   qdmat - query distance matrix  (in/out: may be set here)
 *   qssetypes - query SSE types vector (in/out: may be set here)
 *   qid - query id (in/out: may be set here)
 *   c_qn_addr - address of c_qn device constant (q_qn or c_qn_noshared)
 *   c_qtab_addr - address of c_qtab device constant
 *   c_qdmat_addr - address of c_qdmat device constant
 *   c_qssetypes_addr - address c_qssetypes device constant
 *   
 *
 * Uses the global variables query_dbindex_list, tableaux, etc.
 *
 * Return value: None.
 *
 */
static void copyQueryToConstantMemory(int qi, 
                                      int qn, char *qtab, float *qdmat,
                                      char *qssetypes, char *qid,
                                      int *c_qn_addr,
                                      char *c_qtab_addr,
                                      float *c_qdmat_addr,
                                      char *c_qssetypes_addr)
{
  StopWatchInterface *hTimer = NULL;
  sdkCreateTimer(&hTimer) ;
  sdkResetTimer(&hTimer) ;
  sdkStartTimer(&hTimer) ;
  if (query_dbindex_list)
  {
    int qdbi = query_dbindex_list[qi].index;
    if (query_dbindex_list[qi].large)
    {
      strncpy(qid, large_names+qdbi*(LABELSIZE+1), LABELSIZE);
      // set the qssetypes vector as main diagonal of the query tableau
      for (int i = 0; i < large_orders[qdbi]; i++)
        qssetypes[i] = (large_tableaux+qdbi*MAXDIM*MAXDIM)[INDEX2D(i,i,MAXDIM,MAXDIM)];
      /* copy query structure to constant memory on device */
      checkCudaErrors( hipMemcpy(c_qn_addr, &large_orders[qdbi], sizeof(int), hipMemcpyHostToDevice) );
      /* NB the query in constant memory is MAXDIM not MAXDIM_GPU 
         since constant memory larger than shared memory. */
      checkCudaErrors( hipMemcpy(c_qtab_addr, large_tableaux+qdbi*MAXDIM*MAXDIM, MAXDIM*MAXDIM*sizeof(char), hipMemcpyHostToDevice) );
      checkCudaErrors( hipMemcpy(c_qdmat_addr, large_distmatrices+qdbi*MAXDIM*MAXDIM, MAXDIM*MAXDIM*sizeof(float), hipMemcpyHostToDevice) );
      checkCudaErrors( hipMemcpy(c_qssetypes_addr, qssetypes, MAXDIM*sizeof(char), hipMemcpyHostToDevice) );
    }
    else /* query is in the 'small' structure dbase */
    {
      strncpy(qid, names+qdbi*(LABELSIZE+1), LABELSIZE);
      // set the qssetypes vector as main diagonal of the query tableau
      for (int i = 0; i < orders[qdbi]; i++)
        qssetypes[i] = (tableaux+qdbi*MAXDIM_GPU*MAXDIM_GPU)[INDEX2D(i,i,MAXDIM_GPU,MAXDIM_GPU)];
      /* copy query structure to constant memory on device */
      checkCudaErrors( hipMemcpy(c_qn_addr, &orders[qdbi], sizeof(int), hipMemcpyHostToDevice) );
      /* NB the query in constant memory is MAXDIM not MAXDIM_GPU 
         since constant memory larger than shared memory.
         This means we need to reformat the matrices into the larger 
         size if they are in the smaller class */
      for (int i = 0; i < orders[qdbi]; i++)
      {
        for (int j = i + 1; j < orders[qdbi]; j++)
        {
          char tabcode = (tableaux+qdbi*MAXDIM_GPU*MAXDIM_GPU)[INDEX2D(i,j,MAXDIM_GPU,MAXDIM_GPU)];
          qtab[INDEX2D(i,j,MAXDIM,MAXDIM)] = tabcode;
          qtab[INDEX2D(j,i,MAXDIM,MAXDIM)] = tabcode;
          float dist = (distmatrices+qdbi*MAXDIM_GPU*MAXDIM_GPU)[INDEX2D(i,j,MAXDIM_GPU,MAXDIM_GPU)];
          qdmat[INDEX2D(i,j,MAXDIM,MAXDIM)] = dist;
          qdmat[INDEX2D(j,i,MAXDIM,MAXDIM)] = dist;
        }
      }
      checkCudaErrors( hipMemcpy(c_qtab_addr, qtab, MAXDIM*MAXDIM*sizeof(char), hipMemcpyHostToDevice) );
      checkCudaErrors( hipMemcpy(c_qdmat_addr, qdmat, MAXDIM*MAXDIM*sizeof(float), hipMemcpyHostToDevice) );
      
      checkCudaErrors( hipMemcpy(c_qssetypes_addr, qssetypes, MAXDIM*sizeof(char), hipMemcpyHostToDevice) );
    }
  }
  else // single query mode - copy to constant memory
  {
    fprintf(stderr, "XXX c_qn_addr = %p , qn = %d\n", c_qn_addr,  qn);
    checkCudaErrors( hipMemcpy(c_qn_addr, &qn, sizeof(qn), hipMemcpyHostToDevice) );
    checkCudaErrors( hipMemcpy(c_qtab_addr, qtab, MAXDIM*MAXDIM*sizeof(char), hipMemcpyHostToDevice) );
    checkCudaErrors( hipMemcpy(c_qdmat_addr, qdmat, MAXDIM*MAXDIM*sizeof(float), hipMemcpyHostToDevice) );
    checkCudaErrors( hipMemcpy(c_qssetypes_addr, qssetypes, MAXDIM*sizeof(char), hipMemcpyHostToDevice) );
  }

  sdkStopTimer(&hTimer) ;
  float qtime = sdkGetTimerValue(&hTimer);
  fprintf(stderr, "Copying query to constant memory took %f ms\n", 
          qtime);
}


static void usage(const char *progname)
{
  fprintf(stderr, "Usage: %s [-c] [-q dbfile]\n", progname);
  fprintf(stderr, "  -c : run on host CPU not GPU card\n");
  fprintf(stderr, "  -q dbfile : database is read from dbfile, list of query\n"
          "              ids is read from stdin\n");
  fprintf(stderr, "   -r restarts : number of restarts. Default %d\n",
          DEFAULT_MAXSTART);
  exit(1);
}


int main(int argc, char *argv[])
{
//  CUTThread threadID[MAX_THREADS];
//  int num_threads = 0;
  int exit_status = 0;
  char buf[MAX_LINE_LEN];
  char qtab[MAXDIM*MAXDIM];
  float qdmat[MAXDIM*MAXDIM];
  int qn;
  char qid[LABELSIZE+1];
  int ltype=0,lorder=0,lsoln=0;
  char cltype,clorder,clsoln;
  FILE *dbfp;
  StopWatchInterface *hTimer = NULL;
  int total_dbsize, large_dbsize, gpu_dbsize;
  double dbtime,runtime;
  hipPitchedPtr d_tableaux;
  hipPitchedPtr d_distmatrices;
  int *d_orders;
  int *scores = NULL;
  int *ssemaps = NULL;
  int *d_scores;
  int *d_ssemaps;
  hipError_t cuda_errcode;
  int i,j;
  char qssetypes[MAXDIM];
  int c;
  char *queryptr = NULL;
  int num_queries = 0;
  int large_query_count = 0;
  double norm2score, zscore, pvalue;

  while ((c = getopt(argc, argv, "cq:r:")) != -1)
  {
    switch (c)
    {
      case 'c':
        use_gpu = false;
        break;

      case 'q':
        querydbmode = true;
        strncpy(dbfile, optarg, sizeof(dbfile)-1);
        break;

      case 'r':
        maxstart = atoi(optarg);
        break;
        
      default:
        usage(argv[0]);
        break;
    }
  }

  if (querydbmode)
  {
    cltype = 'T'; ltype = 1;
    clorder = 'T'; lorder = 1;
    clsoln = 'F'; lsoln = 0;
    if (!(queryid_list = (char *)malloc(LABELSIZE+1)))
    {
      fprintf(stderr, "malloc queryid_list failed\n");
      exit(1);
    }
    queryptr = queryid_list;
    while (!feof(stdin))
    {
      //fprintf(stderr, "XXX num_queries = %d\n", num_queries);
      if (num_queries > 0)
      {
        if ((!(queryid_list = (char *)realloc(queryid_list, (num_queries+1)*(LABELSIZE+1)))))
        {
          fprintf(stderr, "realloc queryid_list failed\n");
          exit(1);
        }
        queryptr = queryid_list + num_queries*(LABELSIZE+1);
      }
      if (!fgets(buf, MAX_LINE_LEN, stdin))
        break;
      strncpy(queryptr, buf, LABELSIZE);
      queryptr[LABELSIZE-1] = '\0';
      if (queryptr[strlen(queryptr)-1] == '\n')
        queryptr[strlen(queryptr)-1] = '\0';
      //fprintf(stderr, "XXX queryptr = '%s'\n", queryptr);
      queryptr += (LABELSIZE+1);
      num_queries++;
    }
  }
  else /* not querydbmode: read query tableaux+distmatrices on stdin */
  {
    if (fscanf(stdin, "%s\n", dbfile) != 1)
    {
      fprintf(stderr, "ERROR reading dbfilename from stdin\n");
      exit(1);
    }
    if (fscanf(stdin, "%c %c %c\n", &cltype, &clorder, &clsoln) != 3)
    {
      fprintf(stderr, "ERROR reading options from stdin\n");
      exit(1);
    }
    if (cltype == 'T')
      ltype = 1;
    if (clorder == 'T')
      lorder = 1;
    if (clsoln == 'T')
      lsoln = 1;
    
    num_queries = read_queries(stdin, &query_tableaux, &query_distmatrices,
                               &query_orders, &query_names);
    if (num_queries < 0) {
      fprintf(stderr, "ERROR loading query structures from stdin\n");
      exit(1);
    } else if (num_queries == 0) {
      fprintf(stderr, "ERROR: no query structures found on stdin\n");
      exit(1);
    }
    fprintf(stderr, "Read %d query structures\n", num_queries);
  }

  if (!ltype)
  {
    fprintf(stderr, "WARNING: LTYPE is always set to T\n");
    ltype = 1; cltype = 'T';
  }
  
  if (!(dbfp = fopen(dbfile, "r")))
  {
    fprintf(stderr, "ERROR opening db file %s\n", dbfile);
    exit(1);
  }

  fprintf(stderr, "Loading database...\n");
  sdkCreateTimer(&hTimer) ;
  sdkResetTimer(&hTimer) ;
  sdkStartTimer(&hTimer) ;
  total_dbsize = read_database(dbfp, &tableaux, &distmatrices, 
                               &large_tableaux, &large_distmatrices,
                               &orders, &names,
                               &large_orders, &large_names,
                               &large_dbsize);
  fclose(dbfp);
  if (total_dbsize < 0)
  {
    fprintf(stderr, "ERROR loading database\n");
    exit(1);
  }
  gpu_dbsize = total_dbsize - large_dbsize;
  sdkStopTimer(&hTimer) ;
  dbtime = sdkGetTimerValue(&hTimer);
  fprintf(stderr, "Loaded %d db entries (%d order > %d) in %f ms\n", 
          total_dbsize, large_dbsize, MAXDIM_GPU, dbtime);
          

  if (querydbmode)
  {
    /* Convert the list of query sids to list of indices in db for later
       rapid lookup.
       TODO: we should build a hash table rather than this highly 
       inefficient linear search for each query id, but it's only
       done once and db not that big... 
    */
    fprintf(stderr, "Building query index list...\n");
    sdkResetTimer(&hTimer) ;
    sdkStartTimer(&hTimer) ;
    if (!(query_dbindex_list = (dbIndex_t *)malloc(num_queries*sizeof(dbIndex_t))))
    {
      fprintf(stderr, "malloc query_dbindex_list failed\n");
      exit(1);
    }
    for (i = 0; i < num_queries; i++)
    {
      //fprintf(stderr, "zzz %s\n", queryid_list+i*(LABELSIZE+1));  //XXX
      bool found = false;
      for (j = 0; j < gpu_dbsize; j++) /* search 'small' structure dbase */
      {
        if (!strcasecmp(queryid_list+i*(LABELSIZE+1),names+j*(LABELSIZE+1)))
        {
          query_dbindex_list[i].large = false;
          query_dbindex_list[i].index = j;
          found = true;
          break;
        }
      }
      if (!found)
      {
        for (j = 0; j < large_dbsize; j++) /* search 'large' structure dbase*/
        {
          if (!strcasecmp(queryid_list + i*(LABELSIZE+1),
                           large_names + j*(LABELSIZE+1)))
          {
            query_dbindex_list[i].large = true;
            query_dbindex_list[i].index = j;
            large_query_count++;
            found = true;
            break;
          }
        }
      }
      if (!found)
      {
        fprintf(stderr, "ERROR: query %s not found\n", queryid_list+i*(LABELSIZE+1));
        exit(1);
      }
    }
    sdkStopTimer(&hTimer);
    fprintf(stderr, "Built query index (%d queries (%d large)) in %f ms\n",
            num_queries, large_query_count, sdkGetTimerValue(&hTimer));
  }
  else
  {
    query_dbindex_list = NULL;
  }
    
  /* TODO allow multiple GPUs (need one thread for each) */

  if (use_gpu)
  {
/*
    int devnum = cutGetMaxGflopsDeviceId();
    fprintf(stderr, "using max gflops device %d: ", devnum);
*/
    /* If there is a compute capability 2 device ("Fermi"
       architecture) (or higher) then use that, and do NOT use shared
       memory as it is faster to just rely on the new "NVIDIA Parallel
       DataCache (TM)" -- just use global memory for all (small and large)
       structures
    */

    int devnum, deviceCount, gflops,max_gflops=0, sel_devnum;
    checkCudaErrors( hipGetDeviceCount(&deviceCount) );
    if (deviceCount == 0)
    {
      fprintf(stderr, "There is no device supporting CUDA.\n");
      exit(1);
    }
    fprintf(stderr, "found %d CUDA devices\n", deviceCount);
    for (devnum = 0; devnum < deviceCount; devnum++)
    {  
      hipDeviceProp_t deviceProp;
      hipGetDeviceProperties(&deviceProp, devnum);
      if (deviceProp.major >= 2)
      {
        fprintf(stderr,
          "found modern architecture (compute capability %d.%d) device %d: %s\n",
                deviceProp.major, deviceProp.minor, devnum, deviceProp.name);
        sel_devnum = devnum;
        use_shared_memory = true; 
        break;
      }
      else
      {
        gflops = deviceProp.multiProcessorCount * deviceProp.clockRate;
        fprintf(stderr, "device %d: %s\n", devnum,
                deviceProp.name);
        if (gflops > max_gflops)
        {
          max_gflops = gflops;
          sel_devnum = devnum;
          use_shared_memory = true;
        }
      }
    }
    
    fprintf(stderr, "using device %d: ", sel_devnum);
    hipDeviceProp_t deviceProp;
    hipGetDeviceProperties(&deviceProp, sel_devnum);
    fprintf(stderr, "%s\n", deviceProp.name);
    fprintf(stderr, "totalGlobalMem              = %g GB\n"
                    "sharedMemPerBlock           = %g KB\n"
                    "warpSize                    = %d\n"
                    "maxThreadsPerBlock          = %d\n"
                    "clockRate                   = %g MHz\n"
                    "totalConstMem               = %g KB\n"
                    "multiProcessorCount         = %d\n"
                    "maxThreadsPerMultiProcessor = %d\n"
                    "sharedMeMPerMultiprocessor  = %d KB\n"
                    "maxBlocksPerMultiProcessor  = %d\n",
            (double)deviceProp.totalGlobalMem / (1024*1024*1024),
            (double)deviceProp.sharedMemPerBlock / 1024,
            deviceProp.warpSize,
            deviceProp.maxThreadsPerBlock,
            (double)deviceProp.clockRate / 1000,
            (double)deviceProp.totalConstMem / 1024,
            deviceProp.multiProcessorCount,
            deviceProp.maxThreadsPerMultiProcessor,
            (double)deviceProp.sharedMemPerMultiprocessor / 1024,
            deviceProp.maxBlocksPerMultiProcessor);

    hipSetDevice( sel_devnum );
  }


  fprintf(stderr, "maxstart = %d\n", maxstart);

  srand48(1234);

  if (use_gpu)
  {
    /* setup execution configuration parameters */
    /* TODO optimize for different architectures (automatically) */
    const int blocks = 128;
    const int NUM_THREADS = 128;
    dim3 dimGrid(blocks);          // blocks
    dim3 dimBlock(NUM_THREADS);         // threads per block


    fprintf(stderr, "Execution configuration: Grid = (%d,%d,%d) Block = (%d,%d,%d)\n", dimGrid.x,dimGrid.y,dimGrid.z, dimBlock.x,dimBlock.y,dimBlock.z);



    fprintf(stderr, "using shared memory for small db structs: %s\n",
            use_shared_memory ? "YES" : "NO");

    /* first do the 'small' db structures on GPU (with shared memory) */

    fprintf(stderr, "Copying database to device...\n");
    sdkResetTimer(&hTimer) ;
    sdkStartTimer(&hTimer) ;

    hiprandState *devStates;
  /* allocate space on device for random number generator state */
    int rc;
  if ((rc = hipMalloc((void **)&devStates, 
                       blocks*NUM_THREADS*sizeof(hiprandState))) != hipSuccess)
  {
    fprintf(stderr, "hipMalloc devStates failed %d\n", rc);
    exit(1);
  }
  
  /* initialize device random number generator */
  sdkStartTimer(&hTimer) ;
  init_rng<<<dimGrid, dimBlock>>>(devStates);
  if ((rc = hipGetLastError()) != hipSuccess)
  {
    fprintf(stderr, "init_rng kernel error %d\n", rc);
  }
  hipDeviceSynchronize();
  if ((rc = hipGetLastError()) != hipSuccess)
  {
    fprintf(stderr, "init_rng sync error %d\n", rc);
  }
  sdkStopTimer(&hTimer) ;
  fprintf(stderr, "Initialized device RNG with %d states (%d KB) in %f ms\n",
          blocks*NUM_THREADS, 
          blocks*NUM_THREADS*sizeof(hiprandState)/1024,
          sdkGetTimerValue(&hTimer));

    hipExtent tableaux_extent = make_hipExtent(MAXDIM_GPU, MAXDIM_GPU,
                                                 gpu_dbsize);
    checkCudaErrors( hipMalloc3D(&d_tableaux, tableaux_extent) );
    fprintf(stderr, "d_tableaux.pitch == %u xsize == %u ysize == %u\n", d_tableaux.pitch, d_tableaux.xsize, d_tableaux.ysize);

    hipExtent distmatrices_extent = make_hipExtent(MAXDIM_GPU*sizeof(float), MAXDIM_GPU, gpu_dbsize);
    checkCudaErrors( hipMalloc3D(&d_distmatrices, distmatrices_extent) );
    fprintf(stderr, "d_distmatrices.pitch == %u xsize == %u ysize == %u\n", d_distmatrices.pitch, d_distmatrices.xsize, d_distmatrices.ysize);

    checkCudaErrors( hipMalloc((void **)&d_orders, gpu_dbsize*sizeof(int)) );


    hipMemcpy3DParms copyParams = { 0 };
    // srcPtr is tricky: need to give pitch of row, #elements in row,
    // then height, omitting 3rd dimension (doesn't seem to be documented)
    // (I found this info on 28/1/2010 at 
    // http://sites.google.com/site/cudaiap2009/cookbook-1).
    // Note pitch of row on host is just MAXDIM_GPU, we don't need padding here
    copyParams.srcPtr = make_hipPitchedPtr((void*)tableaux, MAXDIM_GPU, MAXDIM_GPU, MAXDIM_GPU);

    fprintf(stderr, "srcPtr.pitch == %u\n", copyParams.srcPtr.pitch);

    copyParams.dstPtr = d_tableaux;
    copyParams.extent = tableaux_extent;
    copyParams.kind = hipMemcpyHostToDevice;
    checkCudaErrors( hipMemcpy3D(&copyParams) );

    
    hipMemcpy3DParms copyParams2 = { 0 };
    copyParams2.srcPtr = make_hipPitchedPtr((void*)distmatrices,
                                             MAXDIM_GPU*sizeof(float),
                                             MAXDIM_GPU, MAXDIM_GPU);
    fprintf(stderr, "distmatrices srcPtr.pitch == %u\n", copyParams2.srcPtr.pitch);
    copyParams2.dstPtr = d_distmatrices;
    copyParams2.extent = distmatrices_extent;
    copyParams2.kind = hipMemcpyHostToDevice;
    checkCudaErrors( hipMemcpy3D(&copyParams2) );

    checkCudaErrors( hipMemcpy(d_orders, orders, gpu_dbsize*sizeof(int),
                              hipMemcpyHostToDevice) );

    sdkStopTimer(&hTimer) ;
    dbtime = sdkGetTimerValue(&hTimer);
    fprintf(stderr, "Copied %d entries to GPU in %f ms\n", gpu_dbsize, dbtime);

    /* allocate space for output */
    checkCudaErrors( hipMalloc((void **)&d_scores, gpu_dbsize*sizeof(int)));
    if (!(scores = (int *)malloc(gpu_dbsize*sizeof(int))))
    {
      fprintf(stderr, "malloc scores failed\n");
      goto bye;
    }
    if (lsoln)
    {
      checkCudaErrors( hipMalloc((void **)&d_ssemaps, gpu_dbsize*MAXDIM*sizeof(int)));
      if (!(ssemaps = (int *)malloc(gpu_dbsize*MAXDIM*sizeof(int))))
      {
        fprintf(stderr, "malloc ssemaps failed\n");
        goto bye;
      }
    }

    const_addr_t const_addr;
    for (int qi = 0; qi < num_queries; qi++)
    {
      if (!querydbmode) {
      strncpy(qid,  query_names+qi*(LABELSIZE+1), LABELSIZE);
      qn = query_orders[qi];
      // set the qssetypes vector as main diagonal of the query tableau
      for (i = 0; i < query_orders[qi]; i++)
        qssetypes[i] = (query_tableaux+qi*MAXDIM*MAXDIM)[INDEX2D(i,i,MAXDIM,MAXDIM)];

      } else {
        qn = orders[qi];
      }
      if (use_shared_memory) {
        get_device_constant_addresses(&const_addr);
        copyQueryToConstantMemory(qi, qn,
                                  querydbmode ? qtab : query_tableaux+qi*MAXDIM*MAXDIM,
                                  querydbmode ? qdmat : query_distmatrices+qi*MAXDIM*MAXDIM,
                                  qssetypes,
                                  qid,
                                  const_addr.c_qn_addr, const_addr.c_qtab_addr,
                                  const_addr.c_qdmat_addr,
                                  const_addr.c_qssetypes_addr);

//      checkCudaErrors( hipMemcpy(const_addr.c_qn_addr, &qn, sizeof(qn), hipMemcpyHostToDevice) ); fprintf(stderr,"qn=%d\n",qn); //XXX

      }
      else {
        get_device_constant_addresses_noshared_small(&const_addr);    
        copyQueryToConstantMemory(qi, qn,
                                  querydbmode ? qtab : query_tableaux+qi*MAXDIM*MAXDIM,
                                  querydbmode ? qdmat : query_distmatrices+qi*MAXDIM*MAXDIM,
                                  qssetypes,
                                  qid,
                                  const_addr.c_qn_noshared_small_addr,
                                  const_addr.c_qtab_noshared_small_addr, 
                                  const_addr.c_qdmat_noshared_small_addr, 
                                  const_addr.c_qssetypes_noshared_small_addr);
      }
        

      printf("# cudaSaTabsearch LTYPE = %c LORDER = %c LSOLN = %c\n",
             cltype, clorder, clsoln);
      printf("# QUERY ID = %-8s\n", qid);
      printf("# DBFILE = %-80s\n", dbfile);


      fprintf(stderr, "Executing simulated annealing tableaux match kernel (%sshared memory) on GPU for qid %s...\n", use_shared_memory ? " " : "no ", qid);
      checkCudaErrors( hipDeviceSynchronize() );

      sdkResetTimer(&hTimer) ;
      sdkStartTimer(&hTimer) ;
      if (use_shared_memory)
      {
        int xxx_qn=-1; checkCudaErrors( hipMemcpy(&xxx_qn, const_addr.c_qn_addr, sizeof(qn), hipMemcpyDeviceToHost) ); fprintf(stderr,"xxx_qn=%d\n",xxx_qn); //XXX

        sa_tabsearch_gpu<<<dimGrid,dimBlock>>>(gpu_dbsize,
                                               lorder, 
                                               lsoln,
                                               maxstart,
                                               d_tableaux, tableaux_extent,
                                               d_orders,
                                               d_distmatrices, distmatrices_extent,
                                               d_scores,
                                               d_ssemaps,
                                               devStates);
      }
      else 
      {
        sa_tabsearch_gpu_noshared_small<<<dimGrid,dimBlock>>>(gpu_dbsize,
                                               lorder, 
                                               lsoln,
                                               maxstart,
                                               d_tableaux, tableaux_extent,
                                               d_orders,
                                               d_distmatrices, distmatrices_extent,
                                               d_scores,
                                                              d_ssemaps,
                                                              devStates);
      }

      cuda_errcode = hipGetLastError();
      if (cuda_errcode != hipSuccess)
      {
        fprintf(stderr, "kernel launch failed: %s\n", hipGetErrorString(cuda_errcode));
        exit_status = 1;
        goto bye;
      }

      checkCudaErrors( hipDeviceSynchronize() );
      sdkStopTimer(&hTimer) ;
      runtime = sdkGetTimerValue(&hTimer);
      fprintf(stderr,  "GPU execution time %f ms\n", runtime);
      fprintf(stderr,  "%f million iterations/sec\n", ((float)gpu_dbsize * ((float)maxstart * (float)MAXITER) / (runtime/1000)) / 1.0e6);
      
      /* Get results from device */
      checkCudaErrors( hipMemcpy(scores, d_scores, gpu_dbsize*sizeof(int), 
                                hipMemcpyDeviceToHost) );
      if (lsoln)
        checkCudaErrors( hipMemcpy(ssemaps, d_ssemaps, 
                                  gpu_dbsize*MAXDIM*sizeof(int),
                                  hipMemcpyDeviceToHost) );

      /* Wait for host thread */
//XXX      cutWaitForThreads(threadID, num_threads);
//XXX      --num_threads;



      /* TODO we could reduce wasted time waiting by running all host 
         (large db) queries in the one thread instead of matching up
         with GPU query in this loop (actuall, more like the other way
         around usually, the GPU ends up idle while host is still runnign since
         the latter is so much slower even though it has very few
         db entries unlike GPU) */

      for (i = 0; i < gpu_dbsize; i++)
      {
/*        printf("%-8s  %d\n", names+i*(LABELSIZE+1), scores[i]); */
        norm2score = norm2(scores[i], qn, orders[i]);
        zscore = z_gumbel(norm2score, gumbel_a, gumbel_b);
        pvalue = pv_gumbel(zscore);
        printf("%-8s %d %g %g %g\n", 
               names+i*(LABELSIZE+1), scores[i], norm2score, zscore, pvalue);
        if (lsoln)
          for (int k = 0; k < qn; k++)
            if (ssemaps[i*MAXDIM + k] >= 0)
              printf("%3d %3d\n", k+1, ssemaps[i*MAXDIM + k]+1);
      }
    }

    checkCudaErrors( hipFree(d_tableaux.ptr) );
    checkCudaErrors( hipFree(d_distmatrices.ptr) );
    checkCudaErrors( hipFree(d_orders) );
    checkCudaErrors( hipFree(d_scores) );
    free(scores); scores = NULL;
    if (lsoln)
    {
      checkCudaErrors( hipFree(d_ssemaps) );
      free(ssemaps); ssemaps = NULL;
    }

    /* now do the 'large' db structures on GPU (not using shared memory) */
    if (large_dbsize > 0)
    {
      fprintf(stderr, "Copying large structure database to device...\n");
      sdkResetTimer(&hTimer) ;
      sdkStartTimer(&hTimer) ;

      tableaux_extent = make_hipExtent(MAXDIM, MAXDIM, large_dbsize);
      checkCudaErrors( hipMalloc3D(&d_tableaux, tableaux_extent) );
      fprintf(stderr, "d_tableaux.pitch == %u xsize == %u ysize == %u\n", d_tableaux.pitch, d_tableaux.xsize, d_tableaux.ysize);

      distmatrices_extent = make_hipExtent(MAXDIM*sizeof(float), MAXDIM, large_dbsize);
      checkCudaErrors( hipMalloc3D(&d_distmatrices, distmatrices_extent) );
      fprintf(stderr, "d_distmatrices.pitch == %u xsize == %u ysize == %u\n", d_distmatrices.pitch, d_distmatrices.xsize, d_distmatrices.ysize);

      checkCudaErrors( hipMalloc((void **)&d_orders, large_dbsize*sizeof(int)) );


      hipMemcpy3DParms copyParamsl = { 0 };
      // srcPtr is tricky: need to give pitch of row, #elements in row,
      // then height, omitting 3rd dimension (doesn't seem to be documented)
      // (I found this info on 28/1/2010 at 
      // http://sites.google.com/site/cudaiap2009/cookbook-1).
      // Note pitch of row on host is just MAXDIM_GPU, we don't need padding here
      copyParamsl.srcPtr = make_hipPitchedPtr((void*)large_tableaux, MAXDIM, MAXDIM, MAXDIM);

      fprintf(stderr, "srcPtr.pitch == %u\n", copyParamsl.srcPtr.pitch);

      copyParamsl.dstPtr = d_tableaux;
      copyParamsl.extent = tableaux_extent;
      copyParamsl.kind = hipMemcpyHostToDevice;
      checkCudaErrors( hipMemcpy3D(&copyParamsl) );


      hipMemcpy3DParms copyParams2l = { 0 };
      copyParams2l.srcPtr = make_hipPitchedPtr((void*)large_distmatrices,
                                                MAXDIM*sizeof(float),
                                                MAXDIM, MAXDIM);
      fprintf(stderr, "distmatrices srcPtr.pitch == %u\n", copyParams2l.srcPtr.pitch);
      copyParams2l.dstPtr = d_distmatrices;
      copyParams2l.extent = distmatrices_extent;
      copyParams2l.kind = hipMemcpyHostToDevice;
      checkCudaErrors( hipMemcpy3D(&copyParams2l) );

      checkCudaErrors( hipMemcpy(d_orders, large_orders, large_dbsize*sizeof(int),
                                hipMemcpyHostToDevice) );

      sdkStopTimer(&hTimer) ;
      dbtime = sdkGetTimerValue(&hTimer);
      fprintf(stderr, "Copied %d large entries to GPU in %f ms\n", large_dbsize, dbtime);

      /* allocate space for output */
      checkCudaErrors( hipMalloc((void **)&d_scores, large_dbsize*sizeof(int)));
      if (!(scores = (int *)malloc(large_dbsize*sizeof(int))))
      {
        fprintf(stderr, "malloc scores failed\n");
        goto bye;
      }
      if (lsoln)
      {
        checkCudaErrors( hipMalloc((void **)&d_ssemaps, large_dbsize*MAXDIM*sizeof(int)));
        if (!(ssemaps = (int *)malloc(large_dbsize*MAXDIM*sizeof(int))))
        {
          fprintf(stderr, "malloc ssemaps failed\n");
          goto bye;
        }
      }

      for (int qi = 0; qi < num_queries; qi++)
      {
        get_device_constant_addresses_noshared(&const_addr);    
        copyQueryToConstantMemory(qi, qn,
                                  querydbmode ? qtab : query_tableaux+qi*MAXDIM*MAXDIM,
                                  querydbmode ? qdmat : query_distmatrices+qi*MAXDIM*MAXDIM,
                                  qssetypes,
                                  qid,
                                  const_addr.c_qn_noshared_addr,
                                  const_addr.c_qtab_noshared_addr, 
                                  const_addr.c_qdmat_noshared_addr,
                                  const_addr.c_qssetypes_noshared_addr);


        printf("# cudaSaTabsearch LTYPE = %c LORDER = %c LSOLN = %c\n",
               cltype, clorder, clsoln);
        printf("# QUERY ID = %-8s\n", qid);
        printf("# DBFILE = %-80s\n", dbfile);



        fprintf(stderr, "Executing simulated annealing tableaux match kernel (no shared memory) on GPU for qid %s...\n",qid);
        checkCudaErrors( hipDeviceSynchronize() );
        sdkResetTimer(&hTimer) ;
        sdkStartTimer(&hTimer) ;

        int xxx_qn_noshared=-1; checkCudaErrors( hipMemcpy(&xxx_qn_noshared, const_addr.c_qn_noshared_addr, sizeof(qn), hipMemcpyDeviceToHost) ); fprintf(stderr,"xxx_qn_noshared=%d\n",xxx_qn_noshared); //XXX
        sa_tabsearch_gpu_noshared<<<dimGrid,dimBlock>>>(large_dbsize,
                                                        lorder, 
                                                        lsoln,
                                                        maxstart,
                                               d_tableaux, tableaux_extent,
                                               d_orders,
                                               d_distmatrices, distmatrices_extent,
                                               d_scores,
                                                        d_ssemaps,
                                                        devStates);
        cuda_errcode = hipGetLastError();
        if (cuda_errcode != hipSuccess)
        {
          fprintf(stderr, "kernel launch failed: %s\n", hipGetErrorString(cuda_errcode));
          exit_status = 1;
          goto bye;
        }

        checkCudaErrors( hipDeviceSynchronize() );
        sdkStopTimer(&hTimer) ;
        runtime = sdkGetTimerValue(&hTimer);
        fprintf(stderr,  "GPU (no shared memory) execution time %f ms\n", runtime);
        fprintf(stderr,  "%f million iterations/sec\n", ((float)large_dbsize * ((float)maxstart * (float)MAXITER) / (runtime/1000)) / 1.0e6);

        /* Get results from device */
        checkCudaErrors( hipMemcpy(scores, d_scores, large_dbsize*sizeof(int), 
                                    hipMemcpyDeviceToHost) );
        if (lsoln)
          checkCudaErrors( hipMemcpy(ssemaps, d_ssemaps, 
                                    large_dbsize * MAXDIM * sizeof(int),
                                    hipMemcpyDeviceToHost) );

        for (i = 0; i < large_dbsize; i++)
        {
/*          printf("%-8s  %d\n", large_names+i*(LABELSIZE+1), scores[i]); */
          norm2score = norm2(scores[i], qn, large_orders[i]);
          zscore = z_gumbel(norm2score, gumbel_a, gumbel_b);
          pvalue = pv_gumbel(zscore);
          printf("%-8s %d %g %g  %g\n", 
                 large_names+i*(LABELSIZE+1), scores[i], norm2score,
                 zscore, pvalue);
          if (lsoln)
            for (int k = 0; k < qn; k++)
              if (ssemaps[i*MAXDIM + k] >= 0)
                printf("%3d %3d\n", k+1, ssemaps[i*MAXDIM + k]+1);
        }
      }
    }
  }
  else
  {
    /* running on host CPU */

    searchParams_t host_params;
    host_params.ltype = ltype;
    host_params.lorder = lorder;
    host_params.lsoln = lsoln;
    host_params.maxstart = maxstart;
    host_params.num_queries = num_queries;
    host_params.single_query_qid = -1;
    host_params.query_dbindex_list = query_dbindex_list;
    host_params.q_tableaux = query_tableaux;
    host_params.q_distmatrices = query_distmatrices;
    host_params.q_orders = query_orders;
    host_params.q_names = query_names;
    host_params.maxdim = MAXDIM_GPU;
    host_params.dbsize = gpu_dbsize;

    /* first do small structure db */
    host_params.tableaux = tableaux;
    host_params.distmatrices = distmatrices;
    host_params.orders = orders;
    host_params.names = names;
    
    tabsearch_host_thread(&host_params);

    /* then large structure db */
    if (large_dbsize > 0)
    {
      host_params.maxdim = MAXDIM;
      host_params.dbsize = large_dbsize;
      host_params.tableaux = large_tableaux;
      host_params.distmatrices = large_distmatrices;
      host_params.orders = large_orders;
      host_params.names = large_names;
      tabsearch_host_thread(&host_params);
    }
  }

bye:
  /* cleanup and exit */
  free(tableaux);
  free(distmatrices);
  free(orders);
  free(names);
  free(scores);
  free(large_tableaux);
  free(large_distmatrices);
  free(large_names);
  free(large_orders);
  if (lsoln)
    free(ssemaps);
  sdkDeleteTimer( &hTimer);
  if (use_gpu)
  {
    if (large_dbsize > 0)
    {
      checkCudaErrors( hipFree(d_tableaux.ptr) );
      checkCudaErrors( hipFree(d_distmatrices.ptr) );
      checkCudaErrors( hipFree(d_orders) );
      checkCudaErrors( hipFree(d_scores) );
      if (lsoln)
        checkCudaErrors( hipFree(d_ssemaps) );
    }
    hipDeviceReset(); /* replaces deprecated hipDeviceReset() */
  }
  exit(exit_status);
}

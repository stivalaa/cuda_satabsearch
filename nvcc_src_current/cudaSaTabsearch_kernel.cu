#include "hip/hip_runtime.h"
/*****************************************************************************
 * 
 * File:    cudaSaTabsearch_kernel.cu
 * Author:  Alex Stivala
 * Created: January 2010
 *
 * $Id: cudaSaTabsearch_kernel.cu 4755 2013-11-20 03:46:36Z astivala $
 *
 * CUDA kernel for simulated annealing tableau matching (discrete).
 * This is a CUDA implemenation of the FORTRAN subroutine TSAMTD.
 * 
 * if CUDA preprocessor symbol is defined, this is the CUDA kernel version.
 * __DEVICE_EMULATION__ may also be defined for this case (nvcc -deviceemu)
 * in which case device emulation mode is being used
 *
 * Otherwise (CUDA symbol not defined), this builds a host (single threaded)
 * version.
 *
 * if DEBUG (in which case either __DEVICE_EMULATION__ must be defined,
 * or CUDA must not be defined),  is defined then verbose stderr output
 * is generated, and various assertions and checks are compiled in.
 *
 * If CUDA5_DEBUG is defined, then debugging via printf() from device
 * is used.
 *
 * If USE_SHARED_MEMORY is defined, then each block copies the tableau
 * and distance matrix it is operating on frmot he db in global memory
 * into the block shared memory and uses it there, to take advantage
 * of faster (but very small) shared memory. Not using this allows
 * larger structures to be used.  Note that, even when this is not
 * defined, the ssetypes and maxscores vectors are kept in shared
 * memory (this does not limit the maximum size of db structures).
 *
 *****************************************************************************/


#undef CUDA5_DEBUG
#undef TESTING

#if defined(CUDA)
//#include <math_functions.h>
#include <hip/hip_runtime_api.h>
#include <hiprand/hiprand_kernel.h>
#endif
#include "saparams.h"
#include "cudaGetDeviceConstantAddresses.h"

#if defined(__DEVICE_EMULATION__) || !defined(CUDA) || defined(CUDA5_DEBUG)
#include <stdio.h>
#include <assert.h>
#include <stdlib.h>
#endif

#if !defined(CUDA)
#include <math.h>
#include <hip/driver_types.h> /* for struct hipPitchedPtr, hipExtent */
#undef __constant__
#define __constant__
#undef __shared__
#define __shared__ static
#undef __global__
#define __global__ 
#undef __device__
#define __device__
#define hiprandState int
#endif

#define EPS 1.1e-7  /* epsilon for making sure rand is < 1.0 */

#if defined(CUDA) 
#if defined(USE_SHARED_MEMORY)
#define MAXDIM_KERNEL MAXDIM_GPU  /*for shared memory,restrict to MAXDIM_GPU*/
#elif defined(SMALL_MAXDIM)
#define MAXDIM_KERNEL MAXDIM_GPU
#else
#define MAXDIM_KERNEL MAXDIM
#endif
#else
#define MAXDIM_KERNEL MAXDIM      /* otherwise, use largest maxdim */
#endif



/*****************************************************************************
 * 
 * __constant__ memory
 *
 * The query tableau and distance matrix is loaded into constant memory.
 * These are MAXDIM not MAXDIM_KERNEL is constant memory is larger than
 * shared memory so not so restircted (at the moment, on e.g. GTX285,
 * constant memory is 64K but shared memory is only 16K per block).
 *
 *****************************************************************************/

/*tricky: we compile the kernel without CUDA defined for host version
 *annd compile three versions of kernel:
 * (1) with shared memory, maxdim restricted to small so fits in shared
 * (2) without shared memory, large maxdim
 * (3) without shared memory, small maxdim (for Fermi architecture, don't
 *     use shared memory at all, cacheing faster, but still want to run
 *     with small maxdim to not waste so much memory and be consistent
 */

#if !defined(CUDA)
int c_qn_host;    // query structure size
char c_qtab_host[MAXDIM*MAXDIM];  // query tableau
float c_qdmat_host[MAXDIM*MAXDIM];  // query distance matrix
char c_qssetypes_host[MAXDIM]; // main diagonal of c_qn
#endif

/* FIXME because of hack having 3 different versions of everything,
   there is too much __constant__ memory here, but if compiled without
   debug acdtually there is no warning/error and it just gets wrong results
   at runtim for the noshared kernel if __constant__ is used, so 
   changed to __device__ for CUDA 5 just to get it to work.
   It doesn't seem to make any different to speed to use __device__ instead
   of __constant__ though so possibly can just leave it as is.
*/
/* __constant__ */ __device__ int c_qn;    // query structure size
/* __constant__ */ __device__ char c_qtab[MAXDIM*MAXDIM];  // query tableau
/* __constant__ */ __device__ float c_qdmat[MAXDIM*MAXDIM];  // query distance matrix
/* __constant__ */ __device__ char c_qssetypes[MAXDIM]; // main diagonal of c_qn

/* __constant__ */ __device__ int c_qn_noshared;    // query structure size
/* __constant__ */ __device__ char c_qtab_noshared[MAXDIM*MAXDIM];  // query tableau
/* __constant__ */ __device__ float c_qdmat_noshared[MAXDIM*MAXDIM];  // query distance matrix
/* __constant__ */ __device__ char c_qssetypes_noshared[MAXDIM]; // main diagonal of c_qn


/* __constant__ */ __device__ int c_qn_noshared_small;    // query structure size
/* __constant__ */ __device__ char c_qtab_noshared_small[MAXDIM*MAXDIM];  // query tableau
/* __constant__ */ __device__ float c_qdmat_noshared_small[MAXDIM*MAXDIM];  // query distance matrix
/* __constant__ */ __device__ char c_qssetypes_noshared_small[MAXDIM]; // main diagonal of c_qn


#if defined(CUDA)
#if defined(USE_SHARED_MEMORY)
void get_device_constant_addresses
#elif defined(SMALL_MAXDIM)
void get_device_constant_addresses_noshared_small
#else
void get_device_constant_addresses_noshared
#endif
(const_addr_t *const_addr)
{
  checkCudaErrors( hipGetSymbolAddress((void **)&const_addr->c_qn_addr, c_qn) );
  checkCudaErrors( hipGetSymbolAddress((void **)&const_addr->c_qtab_addr, c_qtab) );
  checkCudaErrors( hipGetSymbolAddress((void **)&const_addr->c_qdmat_addr, c_qdmat) );
  checkCudaErrors( hipGetSymbolAddress((void **)&const_addr->c_qssetypes_addr, c_qssetypes) );
  
  checkCudaErrors( hipGetSymbolAddress((void **)&const_addr->c_qn_noshared_addr, c_qn_noshared) );
  checkCudaErrors( hipGetSymbolAddress((void **)&const_addr->c_qtab_noshared_addr, c_qtab_noshared) );
  checkCudaErrors( hipGetSymbolAddress((void **)&const_addr->c_qdmat_noshared_addr, c_qdmat_noshared) );
  checkCudaErrors( hipGetSymbolAddress((void **)&const_addr->c_qssetypes_noshared_addr, c_qssetypes_noshared) );

  checkCudaErrors( hipGetSymbolAddress((void **)&const_addr->c_qn_noshared_small_addr, c_qn_noshared_small) );
  checkCudaErrors( hipGetSymbolAddress((void **)&const_addr->c_qtab_noshared_small_addr, c_qtab_noshared_small) );
  checkCudaErrors( hipGetSymbolAddress((void **)&const_addr->c_qdmat_noshared_small_addr, c_qdmat_noshared_small) );
  checkCudaErrors( hipGetSymbolAddress((void **)&const_addr->c_qssetypes_noshared_small_addr, c_qssetypes_noshared_small) );

}
#endif


#if !defined(CUDA)
/* tricky - we redefined these symbols to the host versions are different */
#define c_qn c_qn_host
#define c_qtab c_qtab_host
#define c_qdmat c_qdmat_host
#define c_qssetypes c_qssetypes_host
#define tscord tscord_cpu
#define deltasd deltasd_cpu
#define thinit thinit_cpu
#define randtypeind randtypeind_cpu
#define icopy icopy_cpu
#define tmscord tmscord_cpu
#define debug_dump_tableau debug_dump_tableu_cpu
#define debug_dump_distmatrix debug_dump_distmatrix_cpu
#else
/* nasty hack -- can't seem to be able to get constant memory in a seaparte
   file, so forced to have 3 different versions of the constant for the
   3 kernel versions. */
#if !defined(USE_SHARED_MEMORY)
#if defined(SMALL_MAXDIM)
#define c_qn c_qn_noshared_small
#define c_qtab c_qtab_noshared_small
#define c_qdmat c_qdmat_noshared_small
#define c_qssetypes c_qssetypes_noshared_small
#define tscord tscord_noshared_small
#define deltasd deltasd_noshared_small
#define thinit thinit_noshared_small
#define randtypeind randtypeind_noshared_small
#define icopy icopy_noshared_small
#define tmscord tmscord_noshared_small
#define debug_dump_tableau debug_dump_tableu_noshared_small
#define debug_dump_distmatrix debug_dump_distmatrix_noshared_small
#else
#define c_qn c_qn_noshared
#define c_qtab c_qtab_noshared
#define c_qdmat c_qdmat_noshared
#define c_qssetypes c_qssetypes_noshared
#define tscord tscord_noshared
#define deltasd deltasd_noshared
#define thinit thinit_noshared
#define randtypeind randtypeind_noshared
#define icopy icopy_noshared
#define tmscord tmscord_noshared
#define debug_dump_tableau debug_dump_tableu_noshared
#define debug_dump_distmatrix debug_dump_distmatrix_noshared
#endif
#endif
#endif

/*****************************************************************************
 * 
 * __device__ functions: callable on GPU only, inlined
 *
 *****************************************************************************/


/* Index into 2d m x n array stored in contiguous memory */
//#define INDEX2D(i,j,m,n) ( ((i)*(n) + (j)) )

/* Get char* to (i,j) element of into 2d array A strored in condiguous
 * memory with pitch (CUDA version of Fortran stride or leading
 * dimension). NB we don't index like INDEX2D but return address as char*
 * which must then be cast to appropriate type, since the address/pitch
 * computations in CUDA are always done in units of bytes, so don't want
 * C address computation using size of actual type */
//#define GET2D(A,i,j,pitch,type) ( *((type *)((char *)(A) + (i)*(pitch) + (j)*sizeof(type) )) )
#define GET2D(A,i,j,pitch,type)  ( ((type *)((A) + ((i)*(pitch))))[(j)] )


/*
 * debug_dump_tableau() - in debug/evmulation build dump tableau
 * 
 * Parameters:
 * tab  (input) 2d char array
 *        tableau one structure. symmetric
 * tab_pitch (input) size_t
 *       pitch of tab
 * n (input) int
 *     dimension of tableau
 *
 * Return value: 
 *   None.
 */
__device__ void debug_dump_tableau(char *tab, size_t tab_pitch, int n)
{
  int i,j;

  for (i = 0; i < n; i++)
  {
    for (j = 0; j <= i; j++)
    {
      printf("%02X ", GET2D(tab, i, j, tab_pitch, char));
    }   
    printf( "\n");
  }
}
/*
 * debug_dump_distmatrix() - in debug/evmulation build dump distmatrix
 * 
 * Parameters:
 * dmat  (input) 2d float array
 *        SSE distance matrix for one structure. symmetric
 * dmat_pitch (input) size_t
 *       pitch of dmat1
 * n (input) int
 *     dimenstion of distance matrix
 *
 * Return value: 
 *   None.
 */
__device__ void debug_dump_distmatrix(char *dmat, size_t dmat_pitch, int n)
{
  int i,j;

  for (i = 0; i < n; i++)
  {
    for (j = 0; j <= i; j++)
    {
      printf( "%5.3f ", GET2D(dmat, i, j, dmat_pitch, float));
    }   
    printf( "\n");
  }
}



/*
 *
 * tscord - Tableau (discrete) matching score function
 *
 *    Return the tableau matching score between two tableau entries
 *    x and y.
 *    The score is 2 if the tableau entries are equal, 1 if they are
 *    equal in only one position, else -2.
 *
 * Parameters:
 *     x, y - the two two-char tableau codes encoded as 4 bits per char
 *            as per parsetableaux.c
 *
 * Return value:
 *     tableau matching score for x and y
 */
__device__ int tscord(char x, char y)
{
  char xhigh,xlow,yhigh,ylow;
  int score;
  
  xhigh = (x & 0xF0);
  xlow =  (x & 0x0F);
  yhigh = (y & 0xF0);
  ylow =  (y & 0x0F);

  score = ( xhigh == yhigh ? (xlow == ylow ? 2 : 1) :
            (xlow == ylow ? 1 : -2) );
/*
  if (xhigh == yhigh)
  {
    if (xlow == ylow)
      score = 2;
    else
      score = 1;
  }
  else if (xlow == ylow)
    score = 1;
  else
    score = -2;
*/
  return score;
}

/*
 * Compute the score for a given SSE matching between two structures
 * given their tableaux (discrete version), and distnace matrices.
 *
 * The score computed is
 *
 * \sum{i=1,j=1}^{N_A} \sum{j=1,k=1}^{N_B} \zeta(T_{ik},T_{kl}) x_{ik}x{jl}
 *
 * in the QIP formulation where x_{ij} is the binary indicator variable
 * indication SSE i in A matched with SSE j in B. 
 *
 * But actually here we are representing the matching with the ssemap
 * vector so can much more efficiently compute this in only
 * O(N_A^2) with 2 nested loops over the ssemap vector rather than 
 * requring O(N_A^2 N_B^2) with 4 nested loops in the naive implentation
 * of the score computation using indicator variables (required only
 * for using a general purpose QP solver, can do it more efficiently here).
 * 
 * Furthermore, we can actually halve the computation since the tableaux
 * matrices are symmetric by only iterating from k = i .. N_A 
 * inside the outer loop i = 1 .. N_A.
 * 
 * Parameters:
 * tab1  (input) encoded as two 4-bit char code
 *        Tableau for one structure. Symmetric.
 *
 * tab1_pitch (input) size_t
 *         pitch of tab1
 *
 * n1     (input) INTEGER
 *        Dimension of tab1 array.
 *
 * tab2  (input) encoded as two 4-bit char code
 *        Tableau for second structure. Symmetric.
 *
 * tab2_pitch (input) size_t
 *        pitch of tab2
 *
 * n2     (input) INTEGER
 *        Dimension of tab2 matrix.
 *
 * dmat1  (input) 2d float array
 *        SSE distance matrix for one structure. symmetric
 *
 * dmat1_pitch (input) size_t
 *       pitch of dmat1
 *
 * dmat2  (input) 2d float array
 *        SSE distance matrix for second structure. symmetric
 *
 * dmat2_pitch (input) size_t
 *         pitch of dmat2
 *
 * ssemap (input) int vector, dimension(n1)
 *        SSE map vector of dimension n1. Each ssemap(i) is the SSE index
 *        in tab2 that SSE i in tab1 is matched with.
 *
 *
 * Return value:
 *   The tableau matching score for given mapping by ssemap.
 *
 */
__device__ int tmscord(char *tab1, size_t tab1_pitch, int n1,
                       char *tab2, size_t tab2_pitch, int n2,
                       char *dmat1, size_t dmat1_pitch, 
                       char *dmat2, size_t dmat2_pitch,
                       int ssemap[])
{
  int i,j,k,l;
  int score;
  
  score = 0;
  for (i = 0; i < n1; i++)
  {
    for (k = i + 1; k  < n1; k++)
    {
      j = ssemap[i];
      l = ssemap[k];
      /* only add to score when both are mapped to something, and */
      /* diagonal entries are SSE type not angle so don't use them either */
#if defined(__DEVICE_EMULATION__) || (defined(DEBUG) && !defined(CUDA)) || defined(CUDA5_DEBUG)
      assert(j == -1 || i != k && j != l);
#endif
      if (j >= 0 && l >= 0)
      {
        /*
#if defined(__DEVICE_EMULATION__) || (defined(DEBUG) && !defined(CUDA))        
        fprintf(stderr, "%d %d %d %d %X %X\n", i,j,k,l,
                GET2D(tab1,i,k,tab1_pitch,char), 
                GET2D(tab2,j,l,tab2_pitch,char));
        fprintf(stderr, "%d %d %d %d %f %f (%f)\n",i,j,k,l,
                GET2D(dmat1,i,k,dmat1_pitch,float) ,
                GET2D(dmat2,j,l,dmat2_pitch,float) ,
                fabsf(GET2D(dmat1,i,k,dmat1_pitch,float) - GET2D(dmat2,j,l,dmat2_pitch,float)) );
#endif
        */
        /* don't add score when difference between SSE distances
           exceeds threshold */
        if (fabsf(GET2D(dmat1,i,k,dmat1_pitch,float) - GET2D(dmat2,j,l,dmat2_pitch,float)) <= MXSSED)
        {
          score += tscord(GET2D(tab1,i,k,tab1_pitch,char), GET2D(tab2,j,l,tab2_pitch,char));
        }
      }
    }
  }
  return score;
}


/*
 * deltasd - 
 *
 * Compute the difference in score from due to removing a particular
 * matching of two SSEs and replacing it with a new one.
 * We can do this in O(N_A) time rather than the O(N_A^2) required for
 * computing the score from scratch as in tmscord.
 *
 * 
 * Parameters:
 * tab1  (input) encoded as two 4-bit char code
 *        Tableau for one structure. Symmetric.
 *
 * tab1_pitch (input) size_t
 *         pitch of tab1
 *
 * n1     (input) INTEGER
 *        Dimension of tab1 array.
 *
 * tab2  (input) encoded as two 4-bit char code
 *        Tableau for second structure. Symmetric.
 *
 * tab2_pitch (input) size_t
 *        pitch of tab2
 *
 * n2     (input) INTEGER
 *        Dimension of tab2 matrix.
 *
 * dmat1  (input) 2d float array
 *        SSE distance matrix for one structure. symmetric
 *
 * dmat1_pitch (input) size_t
 *       pitch of dmat1
 *
 * dmat2  (input) 2d float array
 *        SSE distance matrix for second structure. symmetric
 *
 * dmat2_pitch (input) size_t
 *         pitch of dmat2
 *
 * ssemap (input) int vector, dimension(n1)
 *        SSE map vector of dimension n1. Each ssemap(i) is the SSE index
 *        in tab2 that SSE i in tab1 is matched with.
 *
 * sse_i (input) int 
 *        SSE in tab1 that is being replaced with a new matchig
 *
 * old_j (input) int
 *        SSE in tab2 of old matching
 *
 * new_j (input) int
 *        SSE in tab2 of new matching (matched to new_i)
 *
 *
 * Return value:
 *   The difference to add to the current score due to replacing
 *    the old_i <-> old_k matching with the new_i <-> new_k matching.
 *
 */
__device__ int deltasd(char *tab1, size_t tab1_pitch, int n1,
                       char *tab2, size_t tab2_pitch, int n2,
                       char *dmat1, size_t dmat1_pitch,
                       char *dmat2, size_t dmat2_pitch,
                       int ssemap[],
                       int sse_i,
                       int old_j, int new_j)
{
  int k,l;
  int delta = 0;
  float dmat1_i_k;

#if defined(__DEVICE_EMULATION__) || (defined(DEBUG) && !defined(CUDA))
  fprintf(stderr,"aaa %d %d %d \n", sse_i, old_j, new_j);
#endif

  for (k = 0; k < n1; k++)
  {
    l = ssemap[k];
    if (l >= 0)
    {
      dmat1_i_k = GET2D(dmat1,sse_i,k,dmat1_pitch,float);
      if (old_j >= 0 && l != old_j && k != sse_i && fabsf(dmat1_i_k - GET2D(dmat2,old_j,l,dmat2_pitch,float)) <= MXSSED)
        delta -= tscord(GET2D(tab1,sse_i,k,tab1_pitch,char), GET2D(tab2,old_j,l,tab2_pitch,char));
      
#if defined(__DEVICE_EMULATION__) || (defined(DEBUG) && !defined(CUDA))
      fprintf(stderr,"yyy %d %d %d %d %d\n", sse_i, old_j, new_j,k,l);
#endif
      if (new_j >= 0 && l != new_j && k != sse_i && fabsf(dmat1_i_k - GET2D(dmat2,new_j,l,dmat2_pitch,float)) <= MXSSED)
        delta += tscord(GET2D(tab1,sse_i,k,tab1_pitch,char), GET2D(tab2,new_j,l,tab2_pitch,char));
    }
  }
  return delta;
}


 /*
  * Build the initial mapping of the two structurs for heruristic
  * tableaux matching algoriths.
  *
  * we make an initial matching where we just go along
  * the sequence set match of same SSEs e.g. if 1st in query is helix,
  * match that to first helix in db struture, and so on.
  * (Unless LTYPE flag not set, then we don't care about SSE types and 
  * just go along sequence of SSEs).
  * Then compute the score.
  *
  * Parameters:
  *
  *
  * ssetypes1 (input) char vector length n1
  *        vector of SSE types in structure 1
  *
  * n1     (input) INTEGER
  *        Dimension of tab1 matrix
  *
  * ssetypes2 (input) char vector length n2
  *        vector of SSE types in structure 2
  *
  * n2     (input) INTEGER
  *        Dimension of tab2 matrix.
  *
  * lorder (input) LOGICAL
  *        if true, penalize matches between SSEs not maintaining sequence
  *        order between the tableaux i.e. if i < k and j >= l for i,k
  *        indices in tab1 and j,l indices in tab2.
  *
  *        
  * ssemap (output) INTEGER vector, dimension (n1)
  *        solution SSE map vector of dimension n1. 
  *        Each ssemap(i) is the SSE index
  *        in tab2 that SSE i in tab1 is matched with.
  *
  * revmap (output) INTEGER vector, dimension(n2)
  *     reverse ssemap: revmap(j) for j index in tab2 is the index i
  *     in tab1 that matches that sse i.e. if ssemap(i) = j then
  *     revmap(j) = ssemap(i) and vice versa, for quick lookup of what
  *     is matched so we can easily check that one-to-one mapping maintained
  *
  * state (input/output) State for RNG
  *
  * Return value:
  *        on exit, status of the computation
  *        =  0 : successful exit
  *        =  1 : cannot setup intial ssemap with both lorder and ltype
  */
__device__ int thinit(char ssetypes1[], int n1,
                      char ssetypes2[], int n2,
                      int lorder,
                      int ssemap[], int revmap[],
                      hiprandState *state)
{
  int i,j;
  int info = 0;
  float randnum;

#if defined(__DEVICE_EMULATION__) || (defined(DEBUG) && !defined(CUDA))
  for(int k = 0; k < n1; k++)
    fprintf(stderr,"%02X ", ssetypes1[k]);
  fprintf(stderr, "\n");
  for(int k = 0; k < n2; k++)
    fprintf(stderr,"%02X ", ssetypes2[k]);
  fprintf(stderr, "\n");
#endif

  /* initialize ssemap to all -1 meaning no match for each sse */
  for (i = 0; i < n1; i++)
    ssemap[i] = -1;
  for (j = 0; j < n2; j++)
    revmap[j] = -1;

  /* initial SSE map set by matching along sequence, only matching SSEs
   * of the saem type if LFTYPE flag is set.
   */
  
  j = 0;
  for (i = 0; i < n1; i++)
  {
#if defined(CUDA)
    randnum = hiprand_uniform(state);
#else
    randnum = drand48();
#endif
#if defined(CUDA) && defined( CUDA5_DEBUG )
  const int tid = blockDim.x * blockIdx.x + threadIdx.x; // thread id
    printf("tid = %d thinit randnum = %f\n", tid ,randnum);
#endif
    if (randnum < INIT_MATCHPROB)
    {
      while (j < n2 && ssetypes1[i] != ssetypes2[j])
        j++;
      if (j >= n2) 
      {
        /* not all SSEs in tab1 are mapped, but that's OK */
        info = 0;
        return info;
      }
      else
      {
        ssemap[i] = j;
        revmap[j] = i;
        j++;
      }
    }
  }
  return info;
}

/*
 * find the index of first SSE of same type in tableaux that is not
 * already mapped or -1 if not found
 *
 * Parameters:
 *
 * ssetypesvec (input) char vector length n
 *        vector of SSE types in structure 
 *
 * n     (input) INTEGER
 *        Dimension of tableaux, legnth of ssetypesvec
 *
 * startind (input) INTEGER
 *        SSE index to start at in tab
 *
 * ssetype (input) CHARACTER*2
 *        SSE type as two charcter string 'xa' etc.
 *
 * smap    (input) INTEGER vector, dimension(n1)
 *         each smap(i) is index in other tableau it is already mapped
 *         to, or 0 for not mapped.
 *
 * endind (input) INTEGER
 *         last SSE index to consider in tab 
 *
 * state (input/output) State for RNG
 */
__device__ int randtypeind(char ssetypesvec[], int n,
                           int startind, char ssetype, int smap[], int endind,
                           hiprandState *state)
{
  int i,indi,rti;
  int indlist[MAXDIM];
  float randnum;
  unsigned int randidx;

  i = startind;
  indi = 0;
  rti = -1;
  for (i = startind; i < endind; i++)
  {

#if defined(__DEVICE_EMULATION__) || (defined(DEBUG) && !defined(CUDA)) || defined(CUDA5_DEBUG)
  assert(i >= 0);
  assert(i < n);
#endif

    if (ssetypesvec[i] == ssetype && smap[i] < 0)
      indlist[indi++] = i;
  }

  if (indi == 1)
    rti = indlist[0];
  else if (indi > 1) 
  {
#if defined(CUDA)
    randnum = hiprand_uniform(state);
#else
    randnum = drand48();
#endif
    randidx = (unsigned int)((randnum - EPS) * indi);
    rti = indlist[randidx];
  }
  return rti;
}


/*
 * integer vector copy y <- x
 *
 */
__device__ void icopy(int n, int x[], int y[])
{
  int i;
  for (i = 0; i < n; i++)
    y[i] = x[i];
}

/*****************************************************************************
 * 
 * __global__ functions: GPU kernels, callable from host
 *
 *****************************************************************************/

/*
 * CUDA GPU kernel for tableau matching using simulated annealing.
 *
 * We make an initial matching where we just go along
 * the sequence set match of same SSEs e.g. if 1st in query is helix,
 * match that to first helix in db struture, and so on.
 * (Unless LTYPE flag not set, then we don't care about SSE types and 
 * just go along sequence of SSEs).
 * Then compute the score.
 *
 * Then we use simulated annealing to improve the score. At each 
 * iteration a random SSE is chosen to be remapped to a random
 * other SSE (obeying constraints that are set) or
 * mapped to no SSE in the other structure.
 *
 *
 * "Embarrasingly parallel" version: just do all the loops in here,
 * each thread does a different database structure.
 * The query tableau and distance matrix is placed in constant memory
 * for faster access (constant memory is cached but very limited size:
 * we certainly can't put the whole db of structures there for instance).
 *
 * Parameters:
 *
 * dbsize (input) INTEGER
 *        number of strucures in database
 *
 * lorder (input) LOGICAL
 *        if true, penalize matches between SSEs not maintaining sequence
 *        order between the tableaux i.e. if i < k and j >= l for i,k
 *        indices in tab1 and j,l indices in tab2.
 *
 * lsoln  (input) LOGICAL
 *         if true, return the SSE mapping for the best solution found.
 *
 * maxtart (input) INTEGER
 *         number of restarts (iteratinos of cooling schedule).
 *         Should be a multiple of blocksize.
 *
 * d_qdmat  (input) float array, dimension (n1,n1)
 *        SSE distance matrix for query structure. symmetric
 *
 * d_qdmat_pitch (input) size_t
 *         pitch of d_qdmat
 *
 * d_tableaux (input) pointer to char arrays, CUDA Pitched Pointer
 *        Pointer to database of tableaux
 *
 * tableaux_extent (input) hipExtent
 *        Extent structure for d_tableaux
 *
 * d_ordrers ( input) pointer to ints
 *        Pointer to database of orders (order of each db tableau)
 *
 * d_distmatcies (input) pitched pointer to float arrays
 *        Pointer to database of distance matrices
 *
 * distmatrices_extent (input) hipExtent
 *       Extent structure for d_distmatrices
 *
 * outscore  (output) INTEGER vector, dimension (dbsize)
 *        scores of matching query with each db structure
 *
 * outssemap (output) INTEGER array, dimension (dbszie, n1)
 *        solution SSE map vector of dimension n1 for each db structure
 *        Each ssemap(d,i) is the SSE index
 *        in dbentry d that SSE i in query is matched with.
 *
 * state - (in/out) state for RNG
 */
#if defined(CUDA)
#if defined(USE_SHARED_MEMORY)
__global__ void sa_tabsearch_gpu
#elif defined(SMALL_MAXDIM)
__global__ void sa_tabsearch_gpu_noshared_small
#else
__global__ void sa_tabsearch_gpu_noshared
#endif
#else
void sa_tabsearch_host
#endif
                                (int dbsize,
                                 int lorder,
                                 int lsoln,
                                 int maxstart,
                                 hipPitchedPtr d_tableaux,
                                 hipExtent tableaux_extent,
                                 int *d_orders,
                                 hipPitchedPtr d_distmatrices,
                                 hipExtent distmatrices_extent,
                                 int *outscore,
                                 int *outssemap,
                                 hiprandState *state)
{
 
  /*
   * 
   * __shared__ memory
   *
   * Each block of threads copies one database tableau and distance matrix
   * from the global memory into shared memory. Each thread in the block
   * runs the simulated annealing schedule (with different RNG) on the quey
   * and this shared tableu+distmatrix, so the 'restarts' are pallelized
   * within the block. 
   *
   * Note the shared memory is very restriced in size (16K) so we can
   * only fit limited size structures in it.
   *
   */
#if defined(USE_SHARED_MEMORY)
  __shared__ char s_tab[MAXDIM_KERNEL*MAXDIM_KERNEL];
  __shared__ float s_dmat[MAXDIM_KERNEL*MAXDIM_KERNEL];
#endif
  __shared__ char s_ssetypes[MAXDIM_KERNEL]; // TODO maybe shouldn't use this in shared
  __shared__ int s_maxscores[128]; // FIXME should be max threads in block
  __shared__ int s_maxscore_threadid;

  /*
   * automatic (register and local) memory
   */    

//  const int THREAD_N = blockDim.x * gridDim.x;  // total number of threads
#if defined(CUDA)
  const int tid = blockDim.x * blockIdx.x + threadIdx.x; // thread id
  const int blockid = blockIdx.x;            // block id
  const int gridDimx = gridDim.x;            // number of blocks in grid
  const int blockDimx  = blockDim.x;         // number of threads in block
  const int threadIdxx = threadIdx.x;        // thread id in the block
  hiprandState localState = state[tid];/* cache state in fast local memory */
#else
  const int tid = 0;
  const int blockid = 0;
  const int gridDimx = 1;
  const int blockDimx = 1;
  const int threadIdxx = 0;
  hiprandState localState = 0;/*unused*/
#endif

#ifdef CUDA5_DEBUG
  if(tid==0)
    printf("c_qn = %d\nc_qn_noshared = %d\nc_qn_noshared_small = %d\n",
           c_qn, c_qn_noshared, c_qn_noshared_small);
#endif

  int revmap[MAXDIM_KERNEL];  /* reverse ssemap: revmap(j) for j index in
                         tab2 is the index i in tab1 that matches that
                         sse i.e. if ssemap(i) = j then revmap(j) =
                         ssemap(i) and vice versa, for quick lookup of
                         what is matched so we can easily check that
                         one-to-one mapping maintained revmap has
                         dimension (n2) */
    
  int bestmap[MAXDIM]; /* best ssemap feound. this has dimenion (n1) */
  int ssemap[MAXDIM];
  int maxscore,score,newscore;
  int iter;
  float temp;
  float randnum;
  int startj,endj,k,oldj,newj;
  int ssei;
  char *tab1;
  char *tab2;
  char *dmat1; // we use char* not float* to do pitched pointer arithmetic
  char *dmat2;
  int n1,n2;
  int restart;
  int dbi;
  //int iState;
  size_t tab1_pitch, tab2_pitch,dmat1_pitch,dmat2_pitch;
  int i,j;
  int blockmaxscore;
  int delta;


#if defined(__DEVICE_EMULATION__)
  fprintf(stderr, "running in device emulation mode\n");
  fprintf(stderr, "sizeof(int) == %d\n", sizeof(int));
#if defined(USE_SHARED_MEMORY)
  fprintf(stderr, "using shared memory\n");
#else
  fprintf(stderr, "NOT using shared memory\n");
#endif
  fprintf(stderr, "MAXDIM_KERNEL = %d\n", MAXDIM_KERNEL);
#endif

#if !defined(CUDA)
  fprintf(stderr, "running on host\n");
#endif
  

  n1 = c_qn;
  tab1 = c_qtab;
  tab1_pitch = MAXDIM; /* NB MAXDIM not MAXDIM_KERNEL, see comments on c_qtab */
  dmat1 = (char*)c_qdmat;
  dmat1_pitch = MAXDIM * sizeof(float);


  // each of the gridDim.x blocks does as many as needed to do whole database
  for (dbi = blockid; dbi < dbsize; dbi += gridDimx) 
  {

    n2 = d_orders[dbi];

    // get the tableau aray for db entry index dbi using pitched pointer
    char *d_tableauxPtr = (char *)d_tableaux.ptr;
    size_t tableauxPitch = d_tableaux.pitch;
    size_t tableauxSlicePitch = tableauxPitch * tableaux_extent.height;
    char *tableauxSlice = d_tableauxPtr + dbi * tableauxSlicePitch;
    tab2 = tableauxSlice;
    tab2_pitch = tableauxPitch;

    // and similarly for distmatrices (2d float arrays)
    char *d_distmatricesPtr = (char *)d_distmatrices.ptr;
    size_t distmatricesPitch = d_distmatrices.pitch;
    size_t distmatricesSlicePitch = distmatricesPitch * distmatrices_extent.height;
    char *distmatricesSlice = d_distmatricesPtr + dbi * distmatricesSlicePitch;
    dmat2 = distmatricesSlice;
    dmat2_pitch = distmatricesPitch;

    
    // set the s_ssetypes vector as main diagonal of this db instance tableau
    // in parallel (each thread in block does one element)
    for (j = threadIdxx; j < n2; j += blockDimx)
      s_ssetypes[j] = GET2D(tab2,j,j,tab2_pitch,char); // use global not shared so no sync required

#if defined(USE_SHARED_MEMORY)
    //
    // parallel copy (each thread in block does as many elements as needed)
    // of the db entry for this block into the shared memory for the block
    // we'll have each thread do one row of the copy (may leave threads idle
    // since likely to have more threads in block than rows in tableau).
    //
    for (i = threadIdxx; i < n2; i += blockDimx)
      for (j = 0; j < n2; j++)
      {
        *(s_tab + i*MAXDIM_KERNEL + j) = GET2D(tab2,i,j,tab2_pitch,char);
        *(s_dmat + i*MAXDIM_KERNEL + j) = GET2D(dmat2,i,j,dmat2_pitch,float);
      }
    tab2_pitch = MAXDIM_KERNEL;  /* pitch is now leading dimension in shared */
    dmat2_pitch = MAXDIM_KERNEL*sizeof(float);
#else
    /* not using shared memory, just point the s_* variables to the 
       global memory */
    char *s_tab = tab2;
    char *s_dmat = dmat2;
#endif

#if defined(CUDA)
    // sync point so all threads have loaded into shared memory
    __syncthreads();
#endif

#if defined(CUDA5_DEBUG)
    if (tid == 0) {
      printf("maxstart = %d lsoln = %d\n", maxstart, lsoln);
      printf( "tab1 (n = %d):\n", n1);
    debug_dump_tableau(tab1, tab1_pitch, n1);
      printf( "dmat1:\n");
    debug_dump_distmatrix(dmat1, dmat1_pitch, n1);
    printf("c_q_ssetypes:");
    for (int l = 0; l < n1; l++)
      printf("%02X ", c_qssetypes[l]);
    printf("\n");
    printf( "tab2 (n = %d):\n", n2);
    debug_dump_tableau(s_tab, tab2_pitch, n2);
      printf( "s_dmat:\n");
    debug_dump_distmatrix((char *)s_dmat, dmat2_pitch, n2);
    printf("s_ssetypes:");
    for (int l = 0; l < n2; l++)
      printf("%02X ", s_ssetypes[l]);
    printf("\n");

    }
#endif

    maxscore = -99999;


    // each of the blockDim.x threads in the block does as many iterations
    // as need to get to maxstart restarts
    for (restart = 0; restart < maxstart; restart += blockDimx)
    {
      /* setup initial mapping */
      thinit(c_qssetypes, n1, s_ssetypes, n2, lorder, ssemap, revmap,
             &localState);

      score =  tmscord(tab1, tab1_pitch, n1, s_tab, tab2_pitch, n2, 
                       dmat1, dmat1_pitch, 
                       (char *)s_dmat, dmat2_pitch,
                       ssemap);
      if (score > maxscore)
      {
        maxscore = score;
        icopy(n1, ssemap, bestmap);
      }

      temp = TEMP0;

      for (iter = 0; iter < MAXITER; iter++)
      {
        /* generate neighbour state by picking random SSE in tab1 and
           moving its mapping to a radnom SSE in tab2, maintaining 
           constraints */
#if defined(CUDA)
        randnum = hiprand_uniform(&localState);
#else
        randnum = drand48();
#endif
        ssei = ((randnum - EPS) * n1);

#if defined (DEBUG) && !defined(CUDA)
//        fprintf(stderr, "xxx %f %d\n", randnum, ssei);
#endif
#if defined(CUDA) && defined(CUDA5_DEBUG)
        if (tid==0)
//          printf( "xxx %f %d\n", randnum, ssei);
#endif


        if (lorder)
        {
          startj = ssemap[ssei];
          k = ssei;
          while (startj < 0 && k >= 0)
          {
            startj = ssemap[k];
            k--;
          }
          if (startj < 0)
            startj = n2;
          if (ssei == n1-1)
            endj = n2;
          else if (ssemap[ssei+1] < 0)
          {
            endj = -1;
            k = 1;
            while (endj == -1 && ssei + k < n1)
            {
              endj = ssemap[ssei + k];
              k++;
            }
          }
          else
            endj = ssemap[ssei+1];
        }
        else
        {
          startj = 0;
          endj = n2;
        }
        newj = randtypeind(s_ssetypes, n2, startj, 
                           c_qssetypes[ssei],
                           revmap, endj, &localState);
#if defined(__DEVICE_EMULATION__) || (defined(DEBUG) && !defined(CUDA)) || defined(CUDA5_DEBUG)
#ifdef CUDA
        if(tid==0)
#endif
        {
          printf("%d %d %d %d %d %d %d\n", tid , restart, iter, ssei, startj, endj, newj);
          printf( "%d ssemap: ", tid);
          for (int q = 0; q < n1; q++)
            printf( "%d ", ssemap[q]);
          printf( "\n");
        }
#endif
        oldj = ssemap[ssei];
        delta = deltasd(tab1, tab1_pitch, n1, s_tab, tab2_pitch, n2,
                        dmat1, dmat1_pitch,
                        (char*)s_dmat, dmat2_pitch,
                        ssemap, ssei, oldj, newj);
//#undef TESTING
#ifdef TESTING 
#if defined(__DEVICE_EMULATION__) || !defined(CUDA) || defined(CUDA5_DEBUG)
        int revnewmap[MAXDIM_KERNEL],ssenewmap[MAXDIM];
        icopy(n1, ssemap, ssenewmap);
        icopy(n2, revmap, revnewmap);
          if (newj > -1)
          {
            ssenewmap[ssei] = newj;
            if (oldj > -1)
              revnewmap[oldj] = -1;
            revnewmap[newj] = ssei;
          }
          else
          {
            /* the SSE was removed from the matching */
            if (oldj > -1)
            {
              revnewmap[ssenewmap[ssei]] = -1;
              revnewmap[oldj] = -1;
            }
            ssenewmap[ssei] = -1;
          }
        int fullscore = tmscord(tab1, tab1_pitch, n1, s_tab, tab2_pitch, n2,
                                dmat1, dmat1_pitch,
                                (char*)s_dmat, dmat2_pitch,
                                ssenewmap);
//        fprintf(stderr, "zzz %d %d %d\n", delta, score+delta,fullscore);
        assert(score + delta == fullscore);
#endif
#endif

        newscore = score + delta;
        if (newscore > maxscore)
        {
          maxscore = newscore;
          if (lsoln)
          {
            icopy(n1, ssemap, bestmap);
            if (newj > -1)
              bestmap[ssei] = newj;
            else
              bestmap[ssei] = -1;
#if defined(__DEVICE_EMULATION__) || (defined(DEBUG) && !defined(CUDA)) 
            fprintf(stderr, "NNN %d %d %d\n", ssei, oldj, newj);
            fprintf(stderr, "%d bestmap: ", tid);
            for (int q = 0; q < n1; q++)
              fprintf(stderr, "%d ", bestmap[q]);
            fprintf(stderr, "\n");
#endif
          }
        }

#if defined(__DEVICE_EMULATION__) || (defined(DEBUG) && !defined(CUDA))
        fprintf(stderr, "%d %d %d %f %d %d %f\n",tid, restart, iter, temp, score, newscore,
                expf((float)delta / temp));
#endif
#if defined(CUDA)
        randnum = hiprand_uniform(&localState);
#else
        randnum = drand48();
#endif
        if (expf((float)delta / temp) > randnum)
        {
          /* accept the move, update ssemap and revmap accordingly */
          score = newscore;
          if (newj > -1)
          {
            ssemap[ssei] = newj;
            if (oldj > -1)
              revmap[oldj] = -1;
            revmap[newj] = ssei;
          }
          else
          {
            /* the SSE was removed from the matching */
            if (oldj > -1)
            {
              revmap[ssemap[ssei]] = -1;
              revmap[oldj] = -1;
            }
            ssemap[ssei] = -1;
          }
        }

        temp *= ALPHA;

      }
    }
    
    s_maxscores[threadIdxx] = maxscore;

#if defined(CUDA)
    // synchronization point: now we need to find max score over each thread in
    // block for that block's db structure.
    __syncthreads();
#endif

    // reduction (MAX) over threads in block to get max score for
    // TODO make this a proper reduction operation instead of a 
    // loop in a single thread
    if (threadIdxx == 0)
    {
      s_maxscore_threadid = 0;
      blockmaxscore = s_maxscores[0];
      for (i = 1; i < blockDimx; i++)
      {
        if (s_maxscores[i] > blockmaxscore)
        {
          blockmaxscore = s_maxscores[i];
          s_maxscore_threadid = i;
        }
      }
#if defined(__DEVICE_EMULATION__) || (defined(DEBUG) && !defined(CUDA))
      fprintf(stderr, "%d says maxscore is %d for %d\n", tid, blockmaxscore,dbi);
#endif
      outscore[dbi] = blockmaxscore;
    }

    if (lsoln)
    {
#if defined(CUDA)
    // synchronization point: need to wait for threadid 0 to have found max
      __syncthreads();
#endif
      // Now that we have the best score and the thread that found it,
      // THAT thread only will put its bestssemap as the output SSE map
      if (threadIdxx == s_maxscore_threadid)
        icopy(n1, bestmap, outssemap + dbi * MAXDIM);
    }
  }
  state[tid] = localState; /* copy back new state from local cache */
}


